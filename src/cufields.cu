#include "hip/hip_runtime.h"
// File: cufields.cu
// Author:Tom Ostler
// Last-modified: 31 Jan 2013 21:31:49
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <iomanip>
#include <fstream>
//cuda headers
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
//local cuda headers
#include "../inc/cufields.h"
#include "../inc/cudadefs.h"
#include "../inc/defines.h"
namespace cufields
{
    //perform the convolution in Fourier space
    __global__ void CFConv(int N,
                           hipfftComplex *CCNxx,
                           hipfftComplex *CCNxy,
                           hipfftComplex *CCNxz,
                           hipfftComplex *CCNyx,
                           hipfftComplex *CCNyy,
                           hipfftComplex *CCNyz,
                           hipfftComplex *CCNzx,
                           hipfftComplex *CCNzy,
                           hipfftComplex *CCNzz,
                           hipfftComplex *CCHx,
                           hipfftComplex *CCHy,
                           hipfftComplex *CCHz,
                           hipfftComplex *CCSx,
                           hipfftComplex *CCSy,
                           hipfftComplex *CCSz
                           )
    {
        const int i = blockDim.x*blockIdx.x + threadIdx.x;
        if(i<N)
        {
            CCHx[i].x = (CCNxx[i].x*CCSx[i].x - CCNxx[i].y*CCSx[i].y + CCNxy[i].x*CCSy[i].x - CCNxy[i].y*CCSy[i].y + CCNxz[i].x*CCSz[i].x - CCNxz[i].y*CCSz[i].y);
            CCHx[i].y = (CCNxx[i].x*CCSx[i].y + CCNxx[i].y*CCSx[i].x + CCNxy[i].x*CCSy[i].y + CCNxy[i].y*CCSy[i].x + CCNxz[i].x*CCSz[i].y + CCNxz[i].y*CCSz[i].x);

            CCHy[i].x = (CCNyx[i].x*CCSx[i].x - CCNyx[i].y*CCSx[i].y + CCNyy[i].x*CCSy[i].x - CCNyy[i].y*CCSy[i].y + CCNyz[i].x*CCSz[i].x - CCNyz[i].y*CCSz[i].y);
            CCHy[i].y = (CCNyx[i].x*CCSx[i].y + CCNyx[i].y*CCSx[i].x + CCNyy[i].x*CCSy[i].y + CCNyy[i].y*CCSy[i].x + CCNyz[i].x*CCSz[i].y + CCNyz[i].y*CCSz[i].x);

            CCHz[i].x = (CCNzx[i].x*CCSx[i].x - CCNzx[i].y*CCSx[i].y + CCNzy[i].x*CCSy[i].x - CCNzy[i].y*CCSy[i].y + CCNzz[i].x*CCSz[i].x - CCNzz[i].y*CCSz[i].y);
            CCHz[i].y = (CCNzx[i].x*CCSx[i].y + CCNzx[i].y*CCSx[i].x + CCNzy[i].x*CCSy[i].y + CCNzy[i].y*CCSy[i].x + CCNzz[i].x*CCSz[i].y + CCNzz[i].y*CCSz[i].x);
        }
    }


    //This needs to be done with a seperate kernel because the size (N)
    //of the zero padded spin arrays is bigger than the number of spins
    __global__ void CCopySpin(int zpN,unsigned int N,double *Cspin,int *Czpsn,hipfftReal *CCSx,hipfftReal *CCSy,hipfftReal *CCSz,hipfftReal *CHrx,hipfftReal *CHry,hipfftReal *CHrz)
    {
        const int i = blockDim.x*blockIdx.x + threadIdx.x;
        if(i<zpN)
		{
			CCSx[i]=0.0;
			CCSy[i]=0.0;
			CCSz[i]=0.0;
			CHrx[i]=0.0;
			CHry[i]=0.0;
			CHrz[i]=0.0;
			//lookup the array value for spin i in the zero pad array
			int lzpsn=Czpsn[i];
			//copy the spin data to the zero padded spin arrays
			//for the fourier transform
			if(lzpsn>=0)
			{
				CCSx[i]=float(Cspin[3*lzpsn]);
				CCSy[i]=float(Cspin[3*lzpsn+1]);
				CCSz[i]=float(Cspin[3*lzpsn+2]);
			}
		}
    }

    __global__ void CCopyFields(int N,int zpN,float *CH,int *Czpsn,hipfftReal *CCHx,hipfftReal *CCHy,hipfftReal *CCHz)
    {
        const int i = blockDim.x*blockIdx.x + threadIdx.x;
        if(i<N)
        {
            int lzpsn=Czpsn[i];
            CH[3*i]=(CCHx[lzpsn])/float(zpN);
            CH[3*i+1]=(CCHy[lzpsn])/float(zpN);
            CH[3*i+2]=(CCHz[lzpsn])/float(zpN);
        }
    }
}
