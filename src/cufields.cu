#include "hip/hip_runtime.h"
// File: cufields.cu
// Author:Tom Ostler
// Last-modified: 02 Oct 2014 14:46:04
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <iomanip>
#include <fstream>
//cuda headers
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
//local cuda headers
#include "../inc/cufields.h"
#include "../inc/cudadefs.h"
#include "../inc/defines.h"
#include "../inc/config.h"
#include "../inc/geom.h"
#include "../inc/llg.h"
namespace cufields
{
    //This stores the interaction matrix dimensions for the lookup and the number
    //of species
    __constant__ unsigned int IMDIMS[7]={0,0,3,3,0,0,0},NUMSPEC=0;
    //The number of k-points and the zero pad size
    __constant__ unsigned int K[3]={0,0,0},ZPDIM[3]={0,0,0},CPLXDIM=0;
    //Reduced timestep
    __constant__ double Crdt;

    void copyConstData()
    {
        FIXOUT(config::Info,"Copying const data with cufields scope to card:" << std::flush);
        hipMemcpyToSymbol(*(&Crdt),&llg::rdt,sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(K),geom::Nk.ptr(),3*sizeof(unsigned int));
        hipMemcpyToSymbol(HIP_SYMBOL(ZPDIM),&geom::zpdim,3*sizeof(unsigned int));
        unsigned int nms=geom::ucm.GetNMS();
        hipMemcpyToSymbol(*(&NUMSPEC),&nms,sizeof(unsigned int));
        hipMemcpyToSymbol(*(&CPLXDIM),&geom::cplxdim,sizeof(unsigned int));
        config::Info << "Done" << std::endl;
    }
    //perform the convolution in Fourier space
    __global__ void CFConv(int N,
                           unsigned int NMS,
                           hipfftComplex *CNk,
                           hipfftComplex *CHk,
                           hipfftComplex *CSk
                           )
    {
        const int i = blockDim.x*blockIdx.x + threadIdx.x;
        if(i<N)
        {
            //the number of threads is the zps (zero pad size). We can then find the coordinate of the
            //fourier space k-point
            const unsigned int kx=i/(ZPDIM[0]*K[0]*ZPDIM[1]*K[1]),ky=i%(ZPDIM[0]*K[0]*ZPDIM[1]*K[1])/(ZPDIM[2]*K[2]),kz=i%(ZPDIM[2]*K[2]);
            for(unsigned int s1 = 0 ; s1 < NMS ; s1++)
            {
                for(unsigned int s2 = 0 ; s2 < NMS ; s2++)
                {
                    for(unsigned int alpha = 0 ; alpha < 3 ; alpha++)
                    {

                        unsigned int hfari=(((s1*3+alpha)*ZPDIM[0]*K[0]+kx)*ZPDIM[1]*K[1]+ky)*ZPDIM[2]*K[2]+kz;
                        for(unsigned int beta = 0 ; beta < 3 ; beta++)
                        {
                            //calculate the interaction matrix array element
                            //from the 7D array lookup
                            //(((((i*dim1+j)*dim2+k)*dim3+l)*dim4+m)*dim5+n)*dim6+o
                            //dim0 = NMS, dim1 = NMS, dim2 = 3, dim3 = 3,
                            //dim4 = ZPDIM[0], dim5 = ZPDIM[1], dim6 = ZPDIM[2]

                            unsigned int Nari=(((((s1*NMS+s2)*3+alpha)*3+beta)*ZPDIM[0]*K[0]+kx)*ZPDIM[1]*K[1]+ky)*ZPDIM[2]*K[2]+kz;

                            //Calculate the field and spin array element (5D lookup)
                            //(((i*dim1+j)*dim2+k)*dim3+l)*dim4+m
                            //dim0 = NMS , dim1 = 3
                            //dim2 = ZPDIM[0], dim3 = ZPDIM[1], dim4 = ZPDIM[2]
                            unsigned int sfari=(((s2*3+beta)*ZPDIM[0]*K[0]+kx)*ZPDIM[1]*K[1]+ky)*ZPDIM[2]*K[2]+kz;
                            CHk[hfari].x += (CNk[Nari].x*CSk[sfari].x - CNk[Nari].y*CSk[sfari].y);
                            CHk[hfari].y += (CNk[Nari].x*CSk[sfari].y + CNk[Nari].y*CSk[sfari].x);
                        }
                    }
                }
            }
        }
    }


    //This needs to be done with a seperate kernel because the size (N)
    //of the zero padded spin arrays is bigger than the number of spins
    __global__ void CCopySpin(int N,double *Cspin,hipfftComplex *CSr,unsigned int *Ckx,unsigned int *Cky,unsigned int *Ckz,unsigned int *Cspec)
    {
        const int i = blockDim.x*blockIdx.x + threadIdx.x;
        if(i<N)
        {
            //For a 5D array lookup we need i,j,k,m,n
            //(((i*dim1+j)*dim2+k)*dim3+l)*dim4+m
            //For the spin arrays the indices correspond to
            // i -> species
            // j -> spin component
            // k -> x-coordinate
            // l -> y-coordinate
            // m -> z-coordinate
            // Here lookup i,k,l,m (call them li,lk,ll,lm
            unsigned int lk=Ckx[i],ll=Cky[i],lm=Ckz[i],li=Cspec[i];
            //loop over the 3 spin coordinates (j)
            for(unsigned int lj = 0 ; lj < 3 ; lj++)
            {

                CSr[(((li*3+lj)*ZPDIM[0]*K[0]+lk)*ZPDIM[1]*K[1]+ll)*ZPDIM[2]*K[2]+lm].x=float(Cspin[3*i+lj]);
                //This can probably be removed
                CSr[(((li*3+lj)*ZPDIM[0]*K[0]+lk)*ZPDIM[1]*K[1]+ll)*ZPDIM[2]*K[2]+lm].y=0.0;
            }
        }
    }

    __global__ void CCopyFields(int N,int zpN,float *CH,hipfftComplex *CHr,unsigned int *Ckx,unsigned int *Cky,unsigned int *Ckz,unsigned int *Cspec)
    {
        const int i = blockDim.x*blockIdx.x + threadIdx.x;
        if(i<N)
        {
            //For a 5D array lookup we need i,j,k,m,n
            //(((i*dim1+j)*dim2+k)*dim3+l)*dim4+m
            //For the spin arrays the indices correspond to
            // i -> species
            // j -> spin component
            // k -> x-coordinate
            // l -> y-coordinate
            // m -> z-coordinate
            // Here lookup i,k,l,m (call them li,lk,ll,lm
            unsigned int lk=Ckx[i],ll=Cky[i],lm=Ckz[i],li=Cspec[i];
            //loop over the 3 spin coordinates (j)
            for(unsigned int lj = 0 ; lj < 3 ; lj++)
            {
                CH[3*i+lj]=(CHr[(((li*3+lj)*ZPDIM[0]*K[0]+lk)*ZPDIM[1]*K[1]+ll)*ZPDIM[2]*K[2]+lm].x)/static_cast<float>(zpN);
            }
        }
    }
    //Cuda Set to Zero 5D Real Space Arrays
    __global__ void CZero5DRSArrays(int N,hipfftComplex *CHr,hipfftComplex *CSr)
    {
        const int i = blockDim.x*blockIdx.x + threadIdx.x;
        if(i<N)
        {
            CHr[i].x=0.0;
            CHr[i].y=0.0;
            CSr[i].x=0.0;
            CSr[i].y=0.0;
        }
    }
    //Cuda Set to Zero 5D Fourier Space Arrays
    __global__ void CZero5DFSArrays(int N,hipfftComplex *CHk,hipfftComplex *CSk)
    {
        const int i = blockDim.x*blockIdx.x + threadIdx.x;
        if(i<N)
        {
            CHk[i].x=0.0;
            CHk[i].y=0.0;
            CSk[i].x=0.0;
            CSk[i].y=0.0;
        }
    }
}
