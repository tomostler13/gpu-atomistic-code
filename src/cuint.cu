#include "hip/hip_runtime.h"
// File: cuint.cu
// Author:Tom Ostler
// Last-modified: 13 Apr 2023 02:07:04 PM
#include "../inc/cufields.h"
#include "../inc/hip/hip_runtime.h"
#include "../inc/config.h"
#include "../inc/spins.h"
#include "../inc/geom.h"
#include "../inc/config.h"
#include "../inc/util.h"
#include "../inc/cuint.h"
#include "../inc/fields.h"
#include "../inc/defines.h"
#include "../inc/cudadefs.h"
#include "../inc/anis.h"
#include "../inc/llg.h"
//Cuda headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
//Library headers
#include <cmath>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <iostream>
namespace cuint
{
    //Reduced timestep
    __constant__ double Crdt;
    __constant__ double CK2perp;
    __constant__ double CK2perpdir[3];
    __constant__ double CK2par;
    __constant__ double CK2pardir[3];
    __constant__ double CK4perp;
    __constant__ double CK4perpdirs[3][3]; 
    __constant__ double CK4par;
    __constant__ double CK4pardirs[3][3];
    __constant__ double Cbasemm;


    void copyConstData()
    {
        FIXOUT(config::Info,"Copying const data with cuint scope to card:" << std::flush);
        hipMemcpyToSymbol(*(&Crdt),&llg::rdt,sizeof(double));
        hipMemcpyToSymbol(*(&CK2par),&anis::k2par,sizeof(double));
        hipMemcpyToSymbol(*(&CK2perp),&anis::k2perp,sizeof(double));
        hipMemcpyToSymbol(*(&CK4par),&anis::k4par,sizeof(double));
        hipMemcpyToSymbol(*(&CK4perp),&anis::k4perp,sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(CK2pardir),anis::k2pardir.ptr(),3*sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(CK2perpdir),anis::k2perpdir.ptr(),3*sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(CK4pardirs),anis::k4pardirs.ptr(),9*sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(CK4perpdirs),anis::k4perpdirs.ptr(),9*sizeof(double));
        double bmm=geom::ucm.GetMu(0)*llg::muB;
        hipMemcpyToSymbol(*(&Cbasemm),&bmm,sizeof(double));
        config::Info << "Done" << std::endl;
    }
    __global__ void CHeun1(int N,double T,double appliedx,double appliedy,double appliedz,float *CH,double *Cspin,double *Cespin,float *Crand,double *Cfn,double *Csigma,double *Cllgpf,double *Clambda,double *Ck1u,double *Ck1udir,double *CHstg)
    {
        register const int i = blockDim.x*blockIdx.x + threadIdx.x;
        if(i<N)
        {
            //The prefactor for the thermal term
            const double TP=sqrt(T)*Csigma[i];
            const double llgpf=Cllgpf[i];
            const double lambda=Clambda[i];
            const double lrn[3]={double(Crand[3*i])*TP,double(Crand[3*i+1])*TP,double(Crand[3*i+2])*TP};

            double h[3]={double(CH[3*i])+CHstg[3*i]+lrn[0]+appliedx,double(CH[3*i+1])+CHstg[3*i+1]+lrn[1]+appliedy,double(CH[3*i+2])+CHstg[3*i+2]+lrn[2]+appliedz};
            //printf("%4.5f\t%4.5f\t%4.5f\n",CHstg[3*i],CHstg[3*i+1],CHstg[3*i+2]);

            const double s[3]={Cspin[3*i],Cspin[3*i+1],Cspin[3*i+2]};
            //calculate the field arising from the first order uniaxial anisotropy
            const double k1udir[3]={Ck1udir[3*i],Ck1udir[3*i+1],Ck1udir[3*i+2]};
            const double k1u=Ck1u[i];
            const double sdn = s[0]*k1udir[0] + s[1]*k1udir[1] + s[2]*k1udir[2];

            h[0]+=(k1u*sdn*k1udir[0]);
            h[1]+=(k1u*sdn*k1udir[1]);
            h[2]+=(k1u*sdn*k1udir[2]);

            /*if(i==0)
            {
                printf("%4.10e\t%4.10e\t%4.10e\t%4.10e\n",2.0*CK2perp/Cbasemm,2.0*CK2par/Cbasemm,2.0*CK4perp/Cbasemm,2.0*CK4par/Cbasemm);
                printf("CK2 par %4.10f\t%4.10f\t%4.10f\n",CK2pardir[0],CK2pardir[1],CK2pardir[2]);
                printf("CK2 perp %4.10f\t%4.10f\t%4.10f\n",CK2perpdir[0],CK2perpdir[1],CK2perpdir[2]);
                printf("CK4 par 1 %4.10f\t%4.10f\t%4.10f\n",CK4pardirs[0][0],CK4pardirs[0][1],CK4pardirs[0][2]);
                printf("CK4 par 2 %4.10f\t%4.10f\t%4.10f\n",CK4pardirs[1][0],CK4pardirs[1][1],CK4pardirs[1][2]);
                printf("CK4 par 3 %4.10f\t%4.10f\t%4.10f\n",CK4pardirs[2][0],CK4pardirs[2][1],CK4pardirs[2][2]);
                printf("CK4 perp 1 %4.10f\t%4.10f\t%4.10f\n",CK4perpdirs[0][0],CK4perpdirs[0][1],CK4perpdirs[0][2]);
                printf("CK4 perp 2 %4.10f\t%4.10f\t%4.10f\n",CK4perpdirs[1][0],CK4perpdirs[1][1],CK4perpdirs[1][2]);
                printf("CK4 perp 3 %4.10f\t%4.10f\t%4.10f\n",CK4perpdirs[2][0],CK4perpdirs[2][1],CK4perpdirs[2][2]);
            }*/
            //field from k2_perp
            const double sdk2perpdir=s[0]*CK2perpdir[0]+s[1]*CK2perpdir[1]+s[2]*CK2perpdir[2];
            h[0]+=(2.0*CK2perp/Cbasemm)*sdk2perpdir*CK2perpdir[0];
            h[1]+=(2.0*CK2perp/Cbasemm)*sdk2perpdir*CK2perpdir[1];
            h[2]+=(2.0*CK2perp/Cbasemm)*sdk2perpdir*CK2perpdir[2];
            /*if(i==0)
            {
                printf("CK2perp %4.5f\t%4.5f\t%4.5f\n",(2.0*CK2perp/Cbasemm)*sdk2perpdir*CK2perpdir[0],(2.0*CK2perp/Cbasemm)*sdk2perpdir*CK2perpdir[1],(2.0*CK2perp/Cbasemm)*sdk2perpdir*CK2perpdir[2]);
            }*/
            //field from k2_parallel
            const double sdk2pardir=s[0]*CK2pardir[0]+s[1]*CK2pardir[1]+s[2]*CK2pardir[2];
            h[0]+=(2.0*CK2par/Cbasemm)*sdk2pardir*CK2pardir[0];
            h[1]+=(2.0*CK2par/Cbasemm)*sdk2pardir*CK2pardir[1];
            h[2]+=(2.0*CK2par/Cbasemm)*sdk2pardir*CK2pardir[2];
            /*if(i==0)
            {
                printf("CK2par %4.5f\t%4.5f\t%4.5f\n",(2.0*CK2par/Cbasemm)*sdk2pardir*CK2pardir[0],(2.0*CK2par/Cbasemm)*sdk2pardir*CK2pardir[1],(2.0*CK2par/Cbasemm)*sdk2pardir*CK2pardir[2]);
            }*/

            //field from k4 perp and par
            for(unsigned int dir = 0 ; dir < 3; dir++)
            {
                const double sdk4perpdir=s[0]*CK4perpdirs[dir][0]+s[1]*CK4perpdirs[dir][1]+s[2]*CK4perpdirs[dir][2];
                const double sdk4pardir=s[0]*CK4pardirs[dir][0]+s[1]*CK4pardirs[dir][1]+s[2]*CK4pardirs[dir][2];
                h[0]+=(2.0*CK4perp/Cbasemm)*sdk4perpdir*CK4perpdirs[dir][0];
                h[1]+=(2.0*CK4perp/Cbasemm)*sdk4perpdir*CK4perpdirs[dir][1];
                h[2]+=(2.0*CK4perp/Cbasemm)*sdk4perpdir*CK4perpdirs[dir][2];
                /*if(i==0)
                {
                    printf("CK4perp %4.5f\t%4.5f\t%4.5f\n",(2.0*CK4perp/Cbasemm)*sdk4perpdir*CK4perpdirs[dir][0],(2.0*CK4perp/Cbasemm)*sdk4perpdir*CK4perpdirs[dir][1],(2.0*CK4perp/Cbasemm)*sdk4perpdir*CK4perpdirs[dir][2]);
                }*/


                h[0]+=(2.0*CK4par/Cbasemm)*sdk4pardir*CK4pardirs[dir][0];
                h[1]+=(2.0*CK4par/Cbasemm)*sdk4pardir*CK4pardirs[dir][1];
                h[2]+=(2.0*CK4par/Cbasemm)*sdk4pardir*CK4pardirs[dir][2];
                /*if(i==0)
                {
                    printf("CK4par %4.5f\t%4.5f\t%4.5f\n",(2.0*CK4par/Cbasemm)*sdk4pardir*CK4pardirs[dir][0],(2.0*CK4par/Cbasemm)*sdk4pardir*CK4pardirs[dir][1],(2.0*CK4par/Cbasemm)*sdk4pardir*CK4pardirs[dir][2]);
                }*/
            }


            //printf("%4.10f\t%4.10f\t%4.10f\n",h[0],h[1],h[2]);

            const double sxh[3]={s[1]*h[2] - s[2]*h[1],s[2]*h[0]-s[0]*h[2],s[0]*h[1]-s[1]*h[0]};
            const double sxsxh[3]={s[1]*sxh[2]-s[2]*sxh[1],s[2]*sxh[0]-s[0]*sxh[2],s[0]*sxh[1]-s[1]*sxh[0]};

            double lfn[3]={0,0,0};
            double es[3]={0,0,0};
            double mods=0.0;
            for(unsigned int j = 0 ; j < 3 ; j++)
            {
                lfn[j] = llgpf*(sxh[j]+lambda*sxsxh[j]);
                Cfn[3*i+j]=lfn[j];
                es[j]=s[j]+lfn[j]*Crdt;
                mods+=es[j]*es[j];
            }
            //calculate one over the square root of the spin modulus
            const double nf=rsqrt(mods);
            for(unsigned int j = 0 ; j < 3 ; j++)
            {
                //set the euler spin value and normalize
                Cespin[3*i+j]=es[j]*nf;
            }
        }
    }

    __global__ void CHeun2(int N,double T,double appliedx,double appliedy,double appliedz,float *CH,double *Cspin,double *Cespin,float *Crand,double *Cfn,double *Csigma,double *Cllgpf,double *Clambda,double *Ck1u,double *Ck1udir,double *CDetField,double *CHstg)
    {
        register const int i = blockDim.x*blockIdx.x + threadIdx.x;
        if(i<N)
        {
            //The prefactor for the thermal term
            const double TP=sqrt(T)*Csigma[i];
            const double llgpf=Cllgpf[i];
            const double lambda=Clambda[i];
            const double lrn[3]={double(Crand[3*i])*TP,double(Crand[3*i+1])*TP,double(Crand[3*i+2])*TP};
            double h[3]={double(CH[3*i])+CHstg[3*i]+lrn[0]+appliedx,double(CH[3*i+1])+CHstg[3*i+1]+lrn[1]+appliedy,double(CH[3*i+2])+CHstg[3*i+2]+lrn[2]+appliedz};
/*            if(i==1 || i==0)
            {
                printf("spin=%d\t%4.6f\t%4.6f\t%4.6f\n",i,h[0],h[1],h[2]);
            }*/

            const double s[3]={Cespin[3*i],Cespin[3*i+1],Cespin[3*i+2]};
            //calculate the field arising from the first order uniaxial anisotropy
            const double k1udir[3]={Ck1udir[3*i],Ck1udir[3*i+1],Ck1udir[3*i+2]};
            const double k1u=Ck1u[i];
            const double sdn = s[0]*k1udir[0] + s[1]*k1udir[1] + s[2]*k1udir[2];

            h[0]+=(k1u*sdn*k1udir[0]);
            h[1]+=(k1u*sdn*k1udir[1]);
            h[2]+=(k1u*sdn*k1udir[2]);


            //field from k2_perp
            const double sdk2perpdir=s[0]*CK2perpdir[0]+s[1]*CK2perpdir[1]+s[2]*CK2perpdir[2];
            h[0]+=(2.0*CK2perp/Cbasemm)*sdk2perpdir*CK2perpdir[0];
            h[1]+=(2.0*CK2perp/Cbasemm)*sdk2perpdir*CK2perpdir[1];
            h[2]+=(2.0*CK2perp/Cbasemm)*sdk2perpdir*CK2perpdir[2];
            //field from k2_parallel
            const double sdk2pardir=s[0]*CK2pardir[0]+s[1]*CK2pardir[1]+s[2]*CK2pardir[2];
            h[0]+=(2.0*CK2par/Cbasemm)*sdk2pardir*CK2pardir[0];
            h[1]+=(2.0*CK2par/Cbasemm)*sdk2pardir*CK2pardir[1];
            h[2]+=(2.0*CK2par/Cbasemm)*sdk2pardir*CK2pardir[2];

            //field from k4 perp and par
            for(unsigned int dir = 0 ; dir < 3; dir++)
            {
                const double sdk4perpdir=s[0]*CK4perpdirs[dir][0]+s[1]*CK4perpdirs[dir][1]+s[2]*CK4perpdirs[dir][2];
                const double sdk4pardir=s[0]*CK4pardirs[dir][0]+s[1]*CK4pardirs[dir][1]+s[2]*CK4pardirs[dir][2];
                h[0]+=(2.0*CK4perp/Cbasemm)*sdk4perpdir*CK4perpdirs[dir][0];
                h[1]+=(2.0*CK4perp/Cbasemm)*sdk4perpdir*CK4perpdirs[dir][1];
                h[2]+=(2.0*CK4perp/Cbasemm)*sdk4perpdir*CK4perpdirs[dir][2];

                h[0]+=(2.0*CK4par/Cbasemm)*sdk4pardir*CK4pardirs[dir][0];
                h[1]+=(2.0*CK4par/Cbasemm)*sdk4pardir*CK4pardirs[dir][1];
                h[2]+=(2.0*CK4par/Cbasemm)*sdk4pardir*CK4pardirs[dir][2];
            }



            double ps[3]={Cspin[3*i],Cspin[3*i+1],Cspin[3*i+2]};
            const double sxh[3]={s[1]*h[2] - s[2]*h[1],s[2]*h[0]-s[0]*h[2],s[0]*h[1]-s[1]*h[0]};
            const double sxsxh[3]={s[1]*sxh[2]-s[2]*sxh[1],s[2]*sxh[0]-s[0]*sxh[2],s[0]*sxh[1]-s[1]*sxh[0]};
            const double fn[3]={Cfn[3*i],Cfn[3*i+1],Cfn[3*i+2]};
            double fnp1[3]={0,0,0};
            double mods=0.0;
            for(unsigned int j = 0 ; j < 3 ; j++)
            {
                fnp1[j]=llgpf*(sxh[j]+lambda*sxsxh[j]);
                ps[j]+=(0.5*(fn[j]+fnp1[j])*Crdt);
                mods+=ps[j]*ps[j];
            }
            const double nf=rsqrt(mods);
            for(unsigned int j = 0 ; j < 3 ; j++)
            {
                Cspin[3*i+j]=ps[j]*nf;
                CDetField[3*i+j]=h[j]-lrn[j];
                CH[3*i+j]=0.0;
            }
            //zero the CH array here


        }
    }
}
