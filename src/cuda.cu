#include "hip/hip_runtime.h"
// File: cuda.cu
// Author:Tom Ostler
// Last-modified: 29 Jun 2015 10:05:33
// Formerly cuLLB.cu
#include "../inc/hip/hip_runtime.h"
#include "../inc/config.h"
#include "../inc/spins.h"
#include "../inc/geom.h"
#include "../inc/config.h"
#include "../inc/random.h"
#include "../inc/intmat.h"
#include "../inc/util.h"
#include "../inc/fields.h"
#include "../inc/arrays.h"
#include "../inc/cudadefs.h"
#include "../inc/defines.h"
#include "../inc/cufields.h"
#include "../inc/cuint.h"
#include "../inc/llg.h"
#include "../inc/exch.h"
//Cuda headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
//Library headers
#include <cmath>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <iostream>
namespace cullg
{

    void llgGPU(unsigned int& t)
    {
        //in this case we are using the interaction matrix for both the exchange and the
        //dipole-dipole field so we might aswell update both at once
        if(config::exchm==0)
        {
            cufields::CZero5DRSArrays<<<rsarzpblockspergrid,threadsperblock>>>(geom::zps*3*geom::ucm.GetNMS(),CHr,CSr,CHk,CSk);
            //copy the spin data to the zero padded arrays
            cufields::CCopySpin<<<blockspergrid,threadsperblock>>>(geom::nspins,Cspin,CSr,Ckx,Cky,Ckz,Cspec);
            //forward transform
            spins_forward();
            //perform convolution
            cufields::CFConv<<<zpblockspergrid,threadsperblock>>>(geom::zps,geom::ucm.GetNMS(),CNk,CHk,CSk);
            //transform the fields back
            fields_back();
            //copy the fields from the zero padded array to the demag field array
            cufields::CCopyFields<<<blockspergrid,threadsperblock>>>(geom::nspins,geom::zps,CH,CHr,Ckx,Cky,Ckz,Cspec);
        }
        else if(config::exchm>0 && config::dipm==0 && config::inc_dip==true)
        {
            //calculate the demag field once per spins::update steps and store in CHDemag. This way when we do the
            //matrix multiplication we just add the exchange field to the demag part.
            if(t%spins::update==0)
            {
                cufields::CZero4DRSArrays<<<rsarzpblockspergrid,threadsperblock>>>(geom::zps*3,CHr,CSr,CHk,CSk);
                //copy the spin data to the zero padded arrays
                cufields::CdipCopySpin<<<blockspergrid,threadsperblock>>>(geom::nspins,Cspin,CSr,Ckx,Cky,Ckz,Cmagmom);
                //copy the fields from the zero padded array to the demag field array
//                cufields::CdipCopyFields<<<zpblockspergrid,threadsperblock>>>(geom::nspins,geom::zps,CHDemag,CHr,Ckx,Cky,Ckz);
                //forward transform
                spins_forward();
                cufields::CdipFConv<<<zpblockspergrid,threadsperblock>>>(geom::zps,CNk,CHk,CSk);
                fields_back();
                cufields::CdipCopyFields<<<blockspergrid,threadsperblock>>>(geom::nspins,geom::zps,CHDemag,CHr,Ckx,Cky,Ckz);
                /*
                //copy spin arrays back to CPU
                    float *temp=NULL;
                    temp = new float [3*geom::nspins];
                    CUDA_CALL(hipMemcpy(temp,CHDemag,3*geom::nspins*sizeof(float),hipMemcpyDeviceToHost));
                    for(unsigned int i = 0 ; i < geom::nspins ; i++)
                    {
                        std::cout << i << "\t" << temp[3*i] << "\t" << temp[3*i+1] << "\t" << temp[3*i+2] << std::endl;
                    }
                    exit(0);
                */

            }
            if(config::exchm==1)//DIA
            {
                cufields::CSpMV_DIA<<<blockspergrid,threadsperblock>>>(geom::nspins,Cdiagoffset,Cdxx,Cdyy,Cdzz,Cspin,CHDemag,CH);
                if(config::offdiag)
                {

                }
            }
            else if(config::exchm==2)//CSR
            {
                cufields::CSpMV_CSR<<<blockspergrid,threadsperblock>>>(geom::nspins,Cxadj,Cadjncy,Cdxx,Cdyy,Cdzz,Cspin,CHDemag,CH);
                if(config::offdiag)
                {
                }
            }

        }
        else if(config::exchm>0 && config::inc_dip==false)
        {

            if(config::exchm==1)//DIA
            {
                cufields::CSpMV_DIA<<<blockspergrid,threadsperblock>>>(geom::nspins,Cdiagoffset,Cdxx,Cdyy,Cdzz,Cspin,CHDemag,CH);

                if(config::offdiag)
                {

                }
            }
            else if(config::exchm==2)//CSR
            {
                cufields::CSpMV_CSR<<<blockspergrid,threadsperblock>>>(geom::nspins,Cxadj,Cadjncy,Cdxx,Cdyy,Cdzz,Cspin,CHDemag,CH);
                if(config::offdiag)
                {
                }
            }
        }
        //calcalute the four spin term?
        if(exch::inc4spin)
        {
            cufields::CSpMV_CSR_FourSpin<<<blockspergrid,threadsperblock>>>(geom::nspins,Cxadj_jkl,Cadjncy_j,Cadjncy_k,Cadjncy_l,CH,Cspin);//,CH,Cspin);
        }
        //generate the random numbers
        CURAND_CALL(hiprandGenerateNormal(gen,Crand,3*geom::nspins,0.0,1.0));
/*            float *temp=NULL;
            temp = new float [3*geom::nspins];
            CUDA_CALL(hipMemcpy(temp,CH,3*geom::nspins*sizeof(float),hipMemcpyDeviceToHost));
            for(unsigned int i = 0 ; i < geom::nspins ; i++)
            {
                std::cout << geom::lu(i,0) << "\t" << geom::lu(i,1) << "\t" << geom::lu(i,2) << "\t" << temp[3*i] << "\t" << temp[3*i+1] << "\t" << temp[3*i+2] << std::endl;
            }
            delete [] temp;
            temp=NULL;
            exit(0);*/
        cuint::CHeun1<<<blockspergrid,threadsperblock>>>(geom::nspins,llg::T,llg::applied[0],llg::applied[1],llg::applied[2],CH,Cspin,Cespin,Crand,Cfn,Csigma,Cllgpf,Clambda,Ck1u,Ck1udir);
        //in this case we are using the interaction matrix for both the exchange and the
        //dipole-dipole field so we might aswell update both at once
        if(config::exchm==0)
        {
//            cufields::CZero5DRSArrays<<<rsarzpblockspergrid,threadsperblock>>>(geom::zps*3*geom::ucm.GetNMS(),CHr,CSr,CHk,CSk);
            //copy the spin data to the zero padded arrays
            cufields::CCopySpin<<<blockspergrid,threadsperblock>>>(geom::nspins,Cespin,CSr,Ckx,Cky,Ckz,Cspec);
            //forward transform
            spins_forward();
            //perform convolution
            cufields::CFConv<<<zpblockspergrid,threadsperblock>>>(geom::zps,geom::ucm.GetNMS(),CNk,CHk,CSk);
            //transform the fields back
            fields_back();
            //copy the fields from the zero padded array to the demag field array
            cufields::CCopyFields<<<blockspergrid,threadsperblock>>>(geom::nspins,geom::zps,CH,CHr,Ckx,Cky,Ckz,Cspec);
        }
        else if(config::exchm>0 && config::dipm==0 && config::inc_dip==true)
        {
            //we don't want to update the dipole field here
            if(config::exchm==1)//DIA
            {
                cufields::CSpMV_DIA<<<blockspergrid,threadsperblock>>>(geom::nspins,Cdiagoffset,Cdxx,Cdyy,Cdzz,Cespin,CHDemag,CH);
                if(config::offdiag)
                {

                }
            }
            else if(config::exchm==2)//CSR
            {
                cufields::CSpMV_CSR<<<blockspergrid,threadsperblock>>>(geom::nspins,Cxadj,Cadjncy,Cdxx,Cdyy,Cdzz,Cespin,CHDemag,CH);
                if(config::offdiag)
                {
                }
            }

        }
        else if(config::exchm>0 && config::inc_dip==false)
        {
            if(config::exchm==1)//DIA
            {
                cufields::CSpMV_DIA<<<blockspergrid,threadsperblock>>>(geom::nspins,Cdiagoffset,Cdxx,Cdyy,Cdzz,Cespin,CHDemag,CH);
                if(config::offdiag)
                {

                }
            }
            else if(config::exchm==2)//CSR
            {
                cufields::CSpMV_CSR<<<blockspergrid,threadsperblock>>>(geom::nspins,Cxadj,Cadjncy,Cdxx,Cdyy,Cdzz,Cespin,CHDemag,CH);
                if(config::offdiag)
                {
                }
            }
        }
        //calcalute the four spin term?
        if(exch::inc4spin)
        {
            cufields::CSpMV_CSR_FourSpin<<<blockspergrid,threadsperblock>>>(geom::nspins,Cxadj_jkl,Cadjncy_j,Cadjncy_k,Cadjncy_l,CH,Cespin);
        }
        cuint::CHeun2<<<blockspergrid,threadsperblock>>>(geom::nspins,llg::T,llg::applied[0],llg::applied[1],llg::applied[2],CH,Cspin,Cespin,Crand,Cfn,Csigma,Cllgpf,Clambda,Ck1u,Ck1udir);
        if(t%spins::update==0)
        {
            //copy spin arrays back to CPU
            double *temp=NULL;
            temp = new double [3*geom::nspins];
            CUDA_CALL(hipMemcpy(temp,Cspin,3*geom::nspins*sizeof(double),hipMemcpyDeviceToHost));
            for(unsigned int i = 0 ; i < geom::nspins ; i++)
            {
                spins::Sx[i]=temp[3*i];
                spins::Sy[i]=temp[3*i+1];
                spins::Sz[i]=temp[3*i+2];
                //				std::cout << spins::Sx[i] << "\t" << spins::Sy[i] << "\t" << spins::Sz[i] << "\t" << sqrt(spins::Sx[i]*spins::Sx[i] + spins::Sy[i]*spins::Sy[i] + spins::Sz[i]*spins::Sz[i])<< std::endl;
            }
            delete [] temp;
            temp=NULL;
        }

    }


    void cuinit(int argc,char *argv[])
    {

        config::printline(config::Info);
        config::Info.width(45);config::Info << std::right << "*" << "**CUDA details***" << std::endl;
        FIXOUT(config::Info,"Resetting device:" << std::flush);
        CUDA_CALL(hipDeviceReset());
        SUCCESS(config::Info);

        //the rank of the fourier transform
        try
        {
            config::cfg.readFile(argv[1]);
        }
        catch(const libconfig::FileIOException &fioex)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("I/O error while reading config file");
        }
        catch(const libconfig::ParseException &pex)
        {
            error::errPreamble(__FILE__,__LINE__);
            std::cerr << ". Parse error at " << pex.getFile()  << ":" << pex.getLine() << "-" << pex.getError() << "***\n" << std::endl;
            exit(EXIT_FAILURE);
        }
        libconfig::Setting &setting = config::cfg.lookup("cuda");
        config::Info << std::noshowpos;
        //FIXOUT(config::Info,"NVCC Compiler:" << COMP << std::endl);
        int device_count=0;
        int device=0;
        //---------------------------------------------------------------
        //Get some of the device properties
        if((hipGetDevice(&device))!=hipSuccess)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("Could not get device id");
        }
        if((hipGetDeviceCount(&device_count))!=hipSuccess)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("Could not get number of devices");
        }
        if(device>device_count)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("GPU device greater than count of devices.");
        }
        if(hipGetDeviceProperties(&deviceProp,device)!=hipSuccess)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("Could not get device properties");
        }
        FIXOUT(config::Info,"Selecting device:" << std::endl);
        for(unsigned int i = 0 ; i < device_count ; i++)
        {
            std::stringstream sstr;
            sstr << "Setting device " << i;
            std::string str=sstr.str();
            FIXOUT(config::Info,str << std::flush);
            if(hipSetDevice(device)!=hipSuccess)
            {
                config::Info << "Failed" << std::endl;
            }
            else
            {
                config::Info << "Success" << std::endl;
            }
        }
        FIXOUT(config::Info,"Number of devices:" << device_count << std::endl);
        FIXOUT(config::Info,"Device selected:" << device << std::endl);
        FIXOUT(config::Info,"Device major.minor:" << deviceProp.major << "." << deviceProp.minor << std::endl);
        //---------------------------------------------------------------
        setting.lookupValue("threadsperblock",threadsperblock);
        FIXOUT(config::Info,"Number of threads per block:" << threadsperblock << std::endl);
        blockspergrid=(geom::nspins+threadsperblock-1)/threadsperblock;
        zpblockspergrid=(geom::zps+threadsperblock-1)/threadsperblock;
        //This is the number of block per grid for addressing the elements of the real space
        //spin and field arrays (dimensions: NUMSPEC x 3 x ZPDIM[0] x ZPDIM[1] x ZPDIM[2]
        if(config::exchm==0)
        {
            rsarzpblockspergrid=(geom::zps*3*geom::ucm.GetNMS()+threadsperblock-1)/threadsperblock;
        }
        else if(config::exchm>0)
        {
            rsarzpblockspergrid=(geom::zps*3+threadsperblock-1)/threadsperblock;
        }
        //Same as the rsarzpblockspergrid but with the ZPDIM[2] dimension now replaced with the (ZPDIM[2]+1)/2
        FIXOUT(config::Info,"Blocks per grid:" << blockspergrid << std::endl);
        FIXOUT(config::Info,"Blocks per grid for zero pad workspace:" << zpblockspergrid << std::endl);
        FIXOUT(config::Info,"Blocks per grid for addressing each 5D array:" << rsarzpblockspergrid << std::endl);
        FIXOUT(config::Info,"Device maximum threads per block:" << deviceProp.maxThreadsPerBlock << std::endl);
        FIXOUT(config::Info,"Device registers per block:" << deviceProp.regsPerBlock << std::endl);
        FIXOUT(config::Info,"Device total const memory:" << deviceProp.totalConstMem << " (bytes)" << std::endl);
        FIXOUT(config::Info,"Device total global memory:" << deviceProp.totalGlobalMem << " (bytes)" << std::endl);

        unsigned long long int curandseed=config::seed;
        FIXOUT(config::Info,"Curand seed:" << curandseed << std::endl);
        //initialize the random number generator
        check_cuda_errors(__FILE__,__LINE__);
        FIXOUT(config::Info,"Initializing hiprand random number generator" << std::flush);
        if((hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT))!=HIPRAND_STATUS_SUCCESS)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("CURAND failed to create random number generator");
        }
        check_cuda_errors(__FILE__,__LINE__);
        if((hiprandSetPseudoRandomGeneratorSeed(gen,curandseed))!=HIPRAND_STATUS_SUCCESS)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("CURAND failed to set random number seed");
        }
        check_cuda_errors(__FILE__,__LINE__);

        if((hiprandGenerateSeeds(gen))!=HIPRAND_STATUS_SUCCESS)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("CURAND failed to generate random number generator seeds");
        }
        check_cuda_errors(__FILE__,__LINE__);
        if((cudaThreadSetLimit(hipLimitStackSize,1024))!=hipSuccess)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("CUDA ERROR: Failed to set thread limit");
        }

        config::Info << "Done" << std::endl;
        FIXOUT(config::Info,"Checking for any cuda errors:" << std::flush);
        check_cuda_errors(__FILE__,__LINE__);
        config::Info << "Done" << std::endl;

        FIXOUT(config::Info,"Allocating memory on device" << std::flush);
        allocate_memory_on_card();
        config::Info << "Done" << std::endl;
        FIXOUT(config::Info,"Copying fourier transformed interaction matrix to device:" << std::flush);
        setup_fourier_transform();
        config::Info << "Done" << std::endl;
        config::printline(config::Info);
        config::Info << "NVIDIA-SMI output:\n" << util::exec("nvidia-smi");
        //__constant__ memory only have .cu scope therefore to use the variables 
        //the variables have to be declared in each .cu file and the variables initialized.
        cuint::copyConstData();
        cufields::copyConstData();
    }
}
