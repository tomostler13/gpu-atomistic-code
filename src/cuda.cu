// File: cuda.cu
// Author:Tom Ostler
// Last-modified: 10 Jan 2013 13:47:49
// Formally cuLLB.cu
#include "../inc/cufields.h"
#include "../inc/hip/hip_runtime.h"
#include "../inc/config.h"
#include "../inc/spins.h"
#include "../inc/mat.h"
#include "../inc/geom.h"
#include "../inc/config.h"
#include "../inc/random.h"
#include "../inc/sim.h"
#include "../inc/intmat.h"
#include "../inc/util.h"
#include "../inc/neigh.h"
#include "../inc/fields.h"
#include "../inc/tdp.h"
#include "../inc/cuint.h"
#include "../inc/array3d.h"
#include "../inc/array.h"
#include "../inc/mat.h"
#include "../inc/mf.h"
//Cuda headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
//Library headers
#include <cmath>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <iostream>
#ifdef DEBUG
#define CUDA_CALL(x) do { if((x) != hipSuccess) {\
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
        exit(EXIT_FAILURE);}} while(0)
#else
#define CUDA_CALL(x) (x)
#endif /*DEBUG*/
#ifdef DEBUG
#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
        exit(EXIT_FAILURE);}} while(0)
#else
#define CURAND_CALL(x) (x)
#endif /*DEBUG*/
#ifdef DEBUG
#define CUFFT_CALL(x) do { if((x) != HIPFFT_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
        exit(EXIT_FAILURE);}} while(0)
#else
#define CUFFT_CALL(x) (x)
#endif /*DEBUG*/
#define FIXOUT(a,b) a.width(75);a << std::left << b;
namespace cullb
{
    hipDeviceProp_t deviceProp;
    hiprandGenerator_t gen;
    bool cullbi=false;
    bool cullbmi=false;
    bool cullbffti=false;
    bool cullbinit=false;
    //number of threads per block and blocks per grid
    int threadsperblock,blockspergrid;
    //same but for the zero padded work spaces
    int zpblockspergrid;
    //rank of the FFT
    int nrank=3;
    //device pointers for Fourier space calculations
    static  hipfftComplex *CCNxx=NULL;
    static  hipfftComplex *CCNxy=NULL;
    static  hipfftComplex *CCNxz=NULL;
    static  hipfftComplex *CCNyx=NULL;
    static  hipfftComplex *CCNyy=NULL;
    static  hipfftComplex *CCNyz=NULL;
    static  hipfftComplex *CCNzx=NULL;
    static  hipfftComplex *CCNzy=NULL;
    static  hipfftComplex *CCNzz=NULL;
    static  hipfftComplex *CCSx=NULL;
    static  hipfftComplex *CCSy=NULL;
    static  hipfftComplex *CCSz=NULL;
    static  hipfftComplex *CCHx=NULL;
    static  hipfftComplex *CCHy=NULL;
    static  hipfftComplex *CCHz=NULL;
    //device pointers
    static  double *Cspin=NULL;
    static  double *Cespin=NULL;
    static  int *Cxadj=NULL;
    static  int *Cadjncy=NULL;
    static  float *Cfspin=NULL;
    static  float *CTemp=NULL;
    static  float *Crand=NULL;
    static  float *Ccoord=NULL;
    static  float *CHDemag=NULL;
    static  int *Clu=NULL;
    static  int *Czpsn=NULL;//The is the zero pad spin number
	static  double *CsurfArea=NULL;
	static  double *CW1pf=NULL;
	static  double *CW2pf=NULL;
	static  double *Csigma=NULL;
    static  double *Cfn=NULL;
    //cufft plans
    hipfftHandle C3DP;

    void initGPU()
    {
        //CUDA_CALL(hipDeviceReset());
        float *nsfa=new float[geom::ss];
        util::fillfloat(geom::ss,tdp::systemp.ptr(),nsfa);
        CUDA_CALL(hipMemcpy(CTemp,nsfa,geom::ss*sizeof(float),hipMemcpyHostToDevice));
        double dtemp[3*geom::ss];
        util::copy3vecto1(geom::ss,spins::sx,spins::sy,spins::sz,dtemp);
        CUDA_CALL(hipMemcpy(Cspin,dtemp,3*geom::ss*sizeof(double),hipMemcpyHostToDevice));
        //copy the contents of the Mean Field arrays to the device
        cuint::copyConstData();
        //allocate the function pointers used to calculate temperature
        //dependent properties on the card
        int lmetype=0,lchipartype=0,lchiperptype=0,lexchstifftype=0;
        int lmefunc=-1,lchiparfunc=-1,lchiperpfunc=-1,lexchstifffunc=-1;
        if(tdp::metype=="function")
        {
            lmetype=0;
            lmefunc=tdp::mefunc;
            if(lmefunc!=0)
            {
                error::errPreamble(__FILE__,__LINE__);
                error::errMessage("Magnetization function type not currently supported on GPU");
            }
        }
        else if(tdp::metype=="meanfield")
        {
            lmetype=-1;
        }
        else
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("Magnetization calculation not supported on GPU");
        }
        if(tdp::chipartype=="function")
        {
            lchipartype=0;
            lchiparfunc=tdp::chiparfunc;
            if(lchiparfunc!=0)
            {
                error::errPreamble(__FILE__,__LINE__);
                error::errMessage("Chi par function type not currently supported on GPU");
            }

        }
        else if(tdp::chipartype=="meanfield")
        {
            lchipartype=-1;

        }
        else
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("Parallel susceptibility calculation not supported on GPU");
        }
        if(tdp::chiperptype=="function")
        {
            lchiperptype=0;
            lchiperpfunc=tdp::chiperpfunc;
            if(lchiperpfunc!=0)
            {
                error::errPreamble(__FILE__,__LINE__);
                error::errMessage("Chi perp function type not currently supported on GPU");
            }
        }
        else if(tdp::chiperptype=="meanfield")
        {
            lchiperptype=-1;

        }
        else
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("Perpendicular susceptibility calculation not supported on GPU");
        }
		if(fields::checkexchange==true && tdp::exchstifftype=="function")
		{
			lexchstifftype=0;
			lexchstifffunc=tdp::exchstifffunc;
			if(lexchstifffunc!=0)
			{
				error::errPreamble(__FILE__,__LINE__);
				error::errMessage("Exchange stiffness function type not currently supported on GPU");
			}
		}
        else if(fields::checkexchange==false)
        {
            lexchstifftype=-1;
        }
		else
		{
			error::errPreamble(__FILE__,__LINE__);
			error::errMessage("Exchange stiffness calculation type not supported on GPU");
		}

        cuint::CSetFunctionPointer<<<threadsperblock,blockspergrid>>>(lmetype,lmefunc,lchipartype,lchiparfunc,lchiperptype,lchiperpfunc,lexchstifftype,lexchstifffunc,fields::anist,fields::ff);

        cullbinit=true;

    }
    void LLBGPU(unsigned int& t)
    {

        assert(geom::gi);
        assert(config::lcf);
        assert(neigh::ni);
        assert(util::ui);
        assert(cullbi);
        assert(cullbmi);
        assert(fields::fi);
        assert(tdp::tdpi);
        if(fields::checkdipolar)
        {
            if(t==0 || t%fields::dfu==0)
            {
                if(fields::dfc=="fft")
                {

                    assert(cullbffti);
                    //copy the spin data to the zero padded arrays
                    cufields::CCopySpin<<<threadsperblock,zpblockspergrid>>>(geom::zps,geom::ss,Cspin,Czpsn,CCSx,CCSy,CCSz);
                    //forward transform
                    spins_forward();
                    //perform convolution
                    cufields::CFConv<<<threadsperblock,zpblockspergrid>>>(geom::zps,CCNxx,CCNxy,CCNxz,CCNyx,CCNyy,CCNyz,CCNzx,CCNzy,CCNzz,CCHx,CCHy,CCHz,CCSx,CCSy,CCSz);
                    //transform the fields back
                    fields_back();
                    //copy the fields from the zero padded array to the demag field array
                    cufields::CCopyFields<<<threadsperblock,blockspergrid>>>(geom::ss,geom::zps,CHDemag,Czpsn,CCHx,CCHy,CCHz);
                }
                else if(fields::dfc=="bruteforce")
                {
                    cufields::CBfDip<<<threadsperblock,blockspergrid>>>(geom::ss,geom::gsV*mat::Ms,CHDemag,Cfspin,Ccoord);
                }
            }
        }
        for(unsigned int i = 0 ; i < fields::dfu ; i++)
        {
            //FOR DEBUGGING THE DIPOLAR FIELD/
            
            float temp[3*geom::ss];
            CUDA_CALL(hipMemcpy(temp,CHDemag,3*geom::ss*sizeof(float),hipMemcpyDeviceToHost));
            for(unsigned int i = 0 ; i < geom::ss ; i++)
            {
                int ijk[3]={geom::lu(i,0),geom::lu(i,1),geom::lu(i,2)};
                std::cout << i << "\t" << ijk[0] << "\t" << ijk[1] << "\t" << ijk[2] << "\t" << temp[3*i] << "\t" << temp[3*i+1] << "\t" << temp[3*i+2] << std::endl;
            }
            exit(0);
            
            if(fields::checkthermal==true)
            {
                //generate the random numbers
                CURAND_CALL(hiprandGenerateNormal(gen,Crand,6*geom::ss,0.0,1.0));
            }
            else
            {
                //or set them to zero. This doesn't need to be called each time.
                cufields::CZeroCrand<<<threadsperblock,blockspergrid>>>(geom::ss,Crand);
            }
            cuint::CHeun1<<<threadsperblock,blockspergrid>>>(geom::ss,float(fields::fH[0]),float(fields::fH[1]),float(fields::fH[2]),mat::Tc,CHDemag,Cfspin,Cspin,Cespin,CTemp,Cxadj,Cadjncy,CsurfArea,Csigma,Crand,Cfn);
/*            check_cuda_errors(__FILE__,__LINE__);
            float temp[geom::ss];
            CUDA_CALL(hipMemcpy(temp,CTemp,geom::ss*sizeof(float),hipMemcpyDeviceToHost));
            for(unsigned int i = 0 ; i < geom::ss ; i++)
            {
                std::cout << tdp::systemp[0]+i*10 << "\t" << temp[i] << std::endl;
            }
            exit(0);*/
            cuint::CHeun2<<<threadsperblock,blockspergrid>>>(geom::ss,float(fields::fH[0]),float(fields::fH[1]),float(fields::fH[2]),mat::Tc,Cfspin,Cspin,CHDemag,CTemp,Crand,Cespin,Cfn,Cxadj,Cadjncy,CsurfArea,Csigma);
        }
        t+=fields::dfu;
        double *temp=NULL;
        temp = new double [3*geom::ss];
        CUDA_CALL(hipMemcpy(temp,Cspin,3*geom::ss*sizeof(double),hipMemcpyDeviceToHost));
        for(unsigned int i = 0 ; i < geom::ss ; i++)
        {
            spins::sx[i]=temp[3*i];
            spins::sy[i]=temp[3*i+1];
            spins::sz[i]=temp[3*i+2];
        }

    }


    void cuinit(int argc,char *argv[])
    {
        assert(spins::si);
        assert(mat::mi);
        assert(geom::gi);
        assert(config::lcf);
        config::printline(config::Info);
        config::Info.width(45);config::Info << std::right << "*" << "**CUDA details***" << std::endl;

        nrank=3;
        try
        {
            config::cfg.readFile(argv[1]);
        }
        catch(const libconfig::FileIOException &fioex)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("I/O error while reading config file");
        }
        catch(const libconfig::ParseException &pex)
        {
            error::errPreamble(__FILE__,__LINE__);
            std::cerr << ". Parse error at " << pex.getFile()  << ":" << pex.getLine() << "-" << pex.getError() << "***\n" << std::endl;
            exit(EXIT_FAILURE);
        }
        libconfig::Setting &setting = config::cfg.lookup("cuda");

        FIXOUT(config::Info,"NVCC Compiler:" << COMP << std::endl);
        int device_count=0;
        int device=0;
        if((hipGetDevice(&device))!=hipSuccess)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("Could not get device id");
        }
        if((hipGetDeviceCount(&device_count))!=hipSuccess)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("Could not get number of devices");
        }
        if(device>device_count)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("GPU device greater than count of devices.");
        }
        if(hipGetDeviceProperties(&deviceProp,device)!=hipSuccess)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("Could not get device properties");
        }
        FIXOUT(config::Info,"Number of devices:" << device_count << std::endl);
        FIXOUT(config::Info,"Device selected:" << device << std::endl);
        FIXOUT(config::Info,"Device major.minor:" << deviceProp.major << "." << deviceProp.minor << std::endl);
        if(deviceProp.major < 2)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("Cuda compute capability of 2.0 or greater is required for the use of function pointers.");
        }
        setting.lookupValue("threadsperblock",threadsperblock);
        FIXOUT(config::Info,"Number of threads per block:" << threadsperblock << std::endl);
        blockspergrid=(geom::ss+threadsperblock-1)/threadsperblock;
        zpblockspergrid=(geom::zps+threadsperblock-1)/threadsperblock;
        FIXOUT(config::Info,"Blocks per grid:" << blockspergrid << std::endl);
        FIXOUT(config::Info,"Blocks per grid for zero pad workspace:" << zpblockspergrid << std::endl);
        FIXOUT(config::Info,"Device maximum threads per block:" << deviceProp.maxThreadsPerBlock << std::endl);
        FIXOUT(config::Info,"Device registers per block:" << deviceProp.regsPerBlock << std::endl);
        FIXOUT(config::Info,"Device total const memory:" << deviceProp.totalConstMem << " (bytes)" << std::endl);
        FIXOUT(config::Info,"Device total global memory:" << deviceProp.totalGlobalMem << " (bytes)" << std::endl);
        if((hipSetDevice(device))!=hipSuccess)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("hipSetDevice returned hipErrorInvalidDevice");
        }
        else
        {
            //double check the device was properly selected
            if(hipGetDevice(&device)!=hipSuccess)
            {
                error::errPreamble(__FILE__,__LINE__);
                error::errMessage("Could not get device on double check");
            }
        }
        //initialize the random number generator
        FIXOUT(config::Info,"Initializing hiprand random number generator" << std::flush);
        unsigned long long int curandseed=config::seed;
        if((hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT))!=HIPRAND_STATUS_SUCCESS)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("CURAND failed to create random number generator");
        }
        if((hiprandSetPseudoRandomGeneratorSeed(gen,curandseed))!=HIPRAND_STATUS_SUCCESS)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("CURAND failed to set random number seed");
        }
        if((hiprandGenerateSeeds(gen))!=HIPRAND_STATUS_SUCCESS)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("CURAND failed to generate random number generator seeds");
        }
        config::Info << "Done" << std::endl;
        FIXOUT(config::Info,"Checking for any cuda errors:" << std::flush);
        check_cuda_errors(__FILE__,__LINE__);
        config::Info << "Done" << std::endl;

        FIXOUT(config::Info,"Allocating memory on device" << std::flush);
        unsigned int memcount=0;
        allocate_memory_on_card(memcount);
        config::Info << "Done" << std::endl;
        FIXOUT(config::Info,"Approximate amount of global memory allocated:" << memcount << " (bytes)" << std::endl);
        if(fields::dfc=="fft")
        {
            FIXOUT(config::Info,"Setting up Fourier transforms on device:" << std::flush);
            setup_fourier_transform();
            config::Info << "Done" << std::endl;
            FIXOUT(config::Info,"Estimated memory for cufft transform:" << double(geom::zps)*double(sizeof(float))*2.0/1024.0/1024.0 << std::endl);
        }
        //size_t f,t;
        //hipCtx_t context;
        //hipCtxCreate(&context,0,device);
        //hipError_t result;

        //result=hipMemGetInfo(&f,&t);
        //FIXOUT(config::Info,"Bytes free on device:" << f << std::endl);
        //FIXOUT(config::Info,"Bytes used on device:" << t << std::endl);
        //This should be asserted before running any kernels
        //so that memory is properly allocated.
        cullbi=true;
    }

    void spins_forward()
    {
        check_cuda_errors(__FILE__,__LINE__);
        assert(cullbmi);
        assert(cullbffti);
        CUFFT_CALL(hipfftExecC2C(C3DP,CCSx,CCSx,HIPFFT_FORWARD));
        CUFFT_CALL(hipfftExecC2C(C3DP,CCSy,CCSy,HIPFFT_FORWARD));
        CUFFT_CALL(hipfftExecC2C(C3DP,CCSz,CCSz,HIPFFT_FORWARD));
    }

    void fields_back()
    {
        assert(cullbmi);
        assert(cullbffti);
        CUFFT_CALL(hipfftExecC2C(C3DP,CCHx,CCHx,HIPFFT_BACKWARD));
        CUFFT_CALL(hipfftExecC2C(C3DP,CCHy,CCHy,HIPFFT_BACKWARD));
        CUFFT_CALL(hipfftExecC2C(C3DP,CCHz,CCHz,HIPFFT_BACKWARD));
    }

    void allocate_memory_on_card(unsigned int& memcount)
    {
        //all of the GPU memory allocations should happen here.
        //--------------------------------------------------------------------------------
        CUDA_CALL(hipMalloc((void**)&CCNxx,geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNxy,geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNxz,geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNyx,geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNyy,geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNyz,geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNzx,geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNzy,geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNzz,geom::zps*sizeof(hipfftComplex)));

        CUDA_CALL(hipMalloc((void**)&CCSx,geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCSy,geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCSz,geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCHx,geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCHy,geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCHz,geom::zps*sizeof(hipfftComplex)));
        memcount+=sizeof(hipfftComplex)*geom::zps*12;
        CUDA_CALL(hipMalloc((void**)&Cspin,3*geom::ss*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Cespin,3*geom::ss*sizeof(double)));
        memcount+=sizeof(double)*3*geom::ss*2;
        CUDA_CALL(hipMalloc((void**)&Cxadj,neigh::xadj.size()*sizeof(int)));
        memcount+=neigh::xadj.size()*sizeof(int);
        CUDA_CALL(hipMalloc((void**)&Cadjncy,neigh::adjncy.size()*sizeof(int)));
        memcount+=neigh::adjncy.size()*sizeof(int);
        CUDA_CALL(hipMalloc((void**)&Cfspin,3*geom::ss*sizeof(float)));
        memcount+=3*geom::ss*sizeof(float);
        CUDA_CALL(hipMalloc((void**)&Crand,6*geom::ss*sizeof(float)));
        memcount+=6*geom::ss*sizeof(float);
        CUDA_CALL(hipMalloc((void**)&CTemp,geom::ss*sizeof(float)));
        memcount+=geom::ss*sizeof(float);
        CUDA_CALL(hipMalloc((void**)&Ccoord,3*geom::ss*sizeof(float)));
        CUDA_CALL(hipMalloc((void**)&CHDemag,3*geom::ss*sizeof(float)));
        memcount+=3*geom::ss*sizeof(float)*2;
        CUDA_CALL(hipMalloc((void**)&Clu,3*geom::ss*sizeof(int)));
        memcount+=3*geom::ss*sizeof(int);
        CUDA_CALL(hipMalloc((void**)&Czpsn,geom::ss*sizeof(int)));
        memcount+=geom::ss*sizeof(int);
		CUDA_CALL(hipMalloc((void**)&CsurfArea,neigh::surfArea.size()*sizeof(double)));
        memcount+=neigh::surfArea.size()*sizeof(double);
		CUDA_CALL(hipMalloc((void**)&CW1pf,geom::ss*sizeof(double)));
		CUDA_CALL(hipMalloc((void**)&CW2pf,geom::ss*sizeof(double)));
		CUDA_CALL(hipMalloc((void**)&Csigma,geom::ss*sizeof(double)));
        memcount+=geom::ss*sizeof(double);
        CUDA_CALL(hipMalloc((void**)&Cfn,3*geom::ss*sizeof(double)));
        memcount+=geom::ss*sizeof(double)*3;

        //--------------------------------------------------------------------------------
        //this section sorts out the copying of the data from the CPU to the card
        //--------------------------------------------------------------------------------
        //declare some arrays for doing copying to card
        //Nspins float array, 3*Nspins float array.
        float *nsfa=new float[geom::ss];
        float *tnsfa=new float[3*geom::ss];
        //Nspins double array, 3*Nspins double array
        double *nsda=new double[geom::ss];
        double *tnsda=new double[3*geom::ss];
        //Nspins int array, 3*Nspins int array
        int *nsia=new int[geom::ss];
        int *tnsia=new int[3*geom::ss];
        //copy spin data to single array
        util::copy3vecto1(geom::ss,spins::sx,spins::sy,spins::sz,tnsda);
        //copy spin data to a floating point array
        util::fillfloat(3*geom::ss,tnsda,tnsfa);
        //copy spin data to card
        CUDA_CALL(hipMemcpy(Cspin,tnsda,3*geom::ss*sizeof(double),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(Cfspin,tnsfa,3*geom::ss*sizeof(float),hipMemcpyHostToDevice));

        //copy temperature data
        util::fillfloat(geom::ss,tdp::systemp.ptr(),nsfa);
        CUDA_CALL(hipMemcpy(CTemp,nsfa,geom::ss*sizeof(float),hipMemcpyHostToDevice));
        //copy neighbourlist data
        CUDA_CALL(hipMemcpy(Cxadj,neigh::xadj.ptr(),neigh::xadj.size()*sizeof(int),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(Cadjncy,neigh::adjncy.ptr(),neigh::adjncy.size()*sizeof(int),hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(CsurfArea,neigh::surfArea.ptr(),neigh::surfArea.size()*sizeof(double),hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(CW1pf,tdp::sysW1pf.ptr(),tdp::sysW1pf.size()*sizeof(double),hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(CW2pf,tdp::sysW2pf.ptr(),tdp::sysW2pf.size()*sizeof(double),hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(Csigma,tdp::sysSigma.ptr(),geom::ss*sizeof(double),hipMemcpyHostToDevice));
        
        //take the integer coordinates and calculate the real space coordinates
        for(unsigned int i = 0 ; i < geom::ss ; i++)
        {
            for(unsigned int j = 0 ; j < 3 ; j++)
            {
                tnsfa[3*i+j]=geom::lu(i,j)*geom::gs[j];
                tnsia[3*i+j]=geom::lu(i,j);
            }
        }
        for(unsigned int i = 0 ; i < geom::ss ; i++)
        {
            nsia[i]=spins::csx.getarrayelement(geom::lu(i,0),geom::lu(i,1),geom::lu(i,2));
        }
        CUDA_CALL(hipMemcpy(Ccoord,tnsfa,3*geom::ss*sizeof(float),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(Clu,tnsia,3*geom::ss*sizeof(int),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(Czpsn,nsia,geom::ss*sizeof(int),hipMemcpyHostToDevice));
        //zero the demag field array
        for(unsigned int i = 0 ; i < 3*geom::ss ; i++){tnsfa[i]=0.0;}CUDA_CALL(hipMemcpy(CHDemag,tnsfa,3*geom::ss*sizeof(float),hipMemcpyHostToDevice));
        //--------------------------------------------------------------------------------


        //make sure we clean up when the program exits
        atexit(deallocate_cuda_memory);
        cullbmi=true;
    }

    void setup_fourier_transform()
    {
        assert(intmat::imi);
        assert(intmat::imft);
        assert(cullbmi);

        //Even though we have 9 interaction matrices, 3 field arrays and
        //3 spin arrays we only need one transform in cufft. This is because
        //we can reuse the plan and alternate the sign depending on whether
        //we have a forward or a back transform
        /*Create a 3D FFT plan. */
        if(hipfftPlan3d(&C3DP,geom::zpdim[0],geom::zpdim[1],geom::zpdim[2],HIPFFT_C2C)!=HIPFFT_SUCCESS)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("CUFFT 3D plan creation failed");
        }

        //At this point we can copy the interaction matrix from the CPU
        //as there is no need to do the determination of the interaction
        //matrix on the card. By asserting imft we are assuring that the
        //fourier transform of the interaction matrix has already been
        //completed.
        if(!intmat::imft)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("You must perform the FT of the interaction matrix.");
        }
        else
        {

            //declare a holder on the heap
            Array3D<fftwf_complex> tempxx,tempxy,tempxz,tempyx,tempyy,tempyz,tempzx,tempzy,tempzz;
            tempxx.resize(geom::zpdim[0],geom::zpdim[1],geom::zpdim[2]);
            tempxy.resize(geom::zpdim[0],geom::zpdim[1],geom::zpdim[2]);
            tempxz.resize(geom::zpdim[0],geom::zpdim[1],geom::zpdim[2]);
            tempyx.resize(geom::zpdim[0],geom::zpdim[1],geom::zpdim[2]);
            tempyy.resize(geom::zpdim[0],geom::zpdim[1],geom::zpdim[2]);
            tempyz.resize(geom::zpdim[0],geom::zpdim[1],geom::zpdim[2]);
            tempzx.resize(geom::zpdim[0],geom::zpdim[1],geom::zpdim[2]);
            tempzy.resize(geom::zpdim[0],geom::zpdim[1],geom::zpdim[2]);
            tempzz.resize(geom::zpdim[0],geom::zpdim[1],geom::zpdim[2]);
            for(unsigned int i = 0 ; i < geom::zpdim[0] ; i++)
            {
                for(unsigned int j = 0 ; j < geom::zpdim[1] ; j++)
                {
                    for(unsigned int k = 0 ; k < geom::zpdim[2] ; k++)
                    {
                        for(unsigned int l = 0 ; l < 2 ; l++)
                        {
                            tempxx(i,j,k)[l]=float(intmat::Nxx(i,j,k)[l]);
                            tempxy(i,j,k)[l]=float(intmat::Nxy(i,j,k)[l]);
                            tempxz(i,j,k)[l]=float(intmat::Nxz(i,j,k)[l]);
                            tempyx(i,j,k)[l]=float(intmat::Nyx(i,j,k)[l]);
                            tempyy(i,j,k)[l]=float(intmat::Nyy(i,j,k)[l]);
                            tempyz(i,j,k)[l]=float(intmat::Nyz(i,j,k)[l]);
                            tempzx(i,j,k)[l]=float(intmat::Nzx(i,j,k)[l]);
                            tempzy(i,j,k)[l]=float(intmat::Nzy(i,j,k)[l]);
                            tempzz(i,j,k)[l]=float(intmat::Nzz(i,j,k)[l]);
                        }
                    }
                }
            }

            CUDA_CALL(hipMemcpy(CCNxx,tempxx.ptr(),geom::zps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(CCNxy,tempxy.ptr(),geom::zps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(CCNxz,tempxz.ptr(),geom::zps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(CCNyx,tempyx.ptr(),geom::zps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(CCNyy,tempyy.ptr(),geom::zps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(CCNyz,tempyz.ptr(),geom::zps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(CCNzx,tempzx.ptr(),geom::zps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(CCNzy,tempzy.ptr(),geom::zps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(CCNzz,tempzz.ptr(),geom::zps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
        }
        check_cuda_errors(__FILE__,__LINE__);
        cullbffti=true;
    }

    void deallocate_cuda_memory()
    {
        config::printline(config::Info);
		config::Info.width(45);config::Info << std::right << "*" << "**EXIT information***" << std::endl;
        FIXOUT(config::Info,"Freeing space on GPU device" << std::flush);
        CUDA_CALL(hipFree(CCNxx));
        CUDA_CALL(hipFree(CCNxy));
        CUDA_CALL(hipFree(CCNxz));
        CUDA_CALL(hipFree(CCNyx));
        CUDA_CALL(hipFree(CCNyy));
        CUDA_CALL(hipFree(CCNyz));
        CUDA_CALL(hipFree(CCNzx));
        CUDA_CALL(hipFree(CCNzy));
        CUDA_CALL(hipFree(CCNzz));
        CUDA_CALL(hipFree(CCSx));
        CUDA_CALL(hipFree(CCSy));
        CUDA_CALL(hipFree(CCSz));
        CUDA_CALL(hipFree(CCHx));
        CUDA_CALL(hipFree(CCHy));
        CUDA_CALL(hipFree(CCHz));
        CUDA_CALL(hipFree(Cspin));
        CUDA_CALL(hipFree(Cespin));
        CUDA_CALL(hipFree(Cfspin));
        CUDA_CALL(hipFree(CTemp));
        CUDA_CALL(hipFree(Crand));
        CUDA_CALL(hipFree(Cxadj));
        CUDA_CALL(hipFree(Cadjncy));
        CUDA_CALL(hipFree(Ccoord));
        CUDA_CALL(hipFree(CHDemag));
        CUDA_CALL(hipFree(Clu));
        CUDA_CALL(hipFree(Czpsn));
		CUDA_CALL(hipFree(CsurfArea));
		CUDA_CALL(hipFree(CW1pf));
		CUDA_CALL(hipFree(CW2pf));
		CUDA_CALL(hipFree(Csigma));
        CUDA_CALL(hipFree(Cfn));
        config::Info << "Done" << std::endl;
    }
}
