// File: cuda.cu
// Author:Tom Ostler
// Created: 26/06/2014
// Last-modified: 18 May 2023 02:51:02 PM
#include "../inc/hip/hip_runtime.h"
#include "../inc/config.h"
#include "../inc/spins.h"
#include "../inc/geom.h"
#include "../inc/config.h"
#include "../inc/random.h"
#include "../inc/intmat.h"
#include "../inc/util.h"
#include "../inc/fields.h"
#include "../inc/arrays.h"
#include "../inc/cudadefs.h"
#include "../inc/defines.h"
#include "../inc/cufields.h"
#include "../inc/cuint.h"
#include "../inc/llg.h"
#include "../inc/anis.h"
#include "../inc/exch.h"
//Cuda headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
//Library headers
#include <cmath>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <iostream>
//The function of this file is to house a number of routines
//that deal with a number of underlying routines, such as
//mallocing/de(m)allocing memory, setting up fft's etc.
// Requires: cullg::cuinit() to be called
namespace cullg
{

    void setup_fourier_transform()
    {
        /*Create a 3D FFT plan. */
        int n[3]={geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::zpdim[2]*geom::Nk[2]};
        int *inembed=n;
        int *onembed=n;
        int istride=1;
        int ostride=1;
        int idist=geom::zps;
        int odist=geom::zps;
        if(config::exchm==0 || (config::dipm==0 && config::inc_dip==true))
        {

            config::openLogFile();
            config::printline(config::Log);
            FIXOUT(config::Log,"Parameters entering into CUFFT plan of the spin arrays (forward)" << std::endl);
            FIXOUTVEC(config::Log,"Dimensions of FFT = ",n[0],n[1],n[2]);
            FIXOUT(config::Log,"rank (dimension of FFT) = " << 3 << std::endl);
            int howmany=0;
            if(config::exchm==0)
            {
                howmany=geom::ucm.GetNMS()*3;
            }
            else if(config::exchm>0 && config::dipm==0 && config::inc_dip)
            {
                howmany=3;
            }
            FIXOUT(config::Log,"How many (FFT's) = " << howmany << std::endl);
            FIXOUTVEC(config::Log,"inembed = ",inembed[0],inembed[1],inembed[2]);
            FIXOUT(config::Log,"istride = " << istride << std::endl);
            FIXOUT(config::Log,"idist = " << idist << std::endl);
            FIXOUTVEC(config::Log,"onembed = ",onembed[0],onembed[1],onembed[2]);
            FIXOUT(config::Log,"ostride = " << ostride << std::endl);
            FIXOUT(config::Log,"odist = " << odist << std::endl);
            FIXOUT(config::Log,"Direction (sign) = " << "HIPFFT_FORWARD" << std::endl);
            if(hipfftPlanMany(&SPc2c,3,n,inembed,istride,idist,onembed,ostride,odist,HIPFFT_C2C,howmany)!=HIPFFT_SUCCESS)
            {
                error::errPreamble(__FILE__,__LINE__);
                error::errMessage("CUFFT 3D plan creation failed");
            }
            else
            {
                FIXOUT(config::Log,"CUFFT returned success");
            }
            config::printline(config::Log);
            FIXOUT(config::Log,"Parameters entering into CUFFT plan of the field arrays (inverse)" << std::endl);
            FIXOUTVEC(config::Log,"Dimensions of FFT = ",n[0],n[1],n[2]);
            FIXOUT(config::Log,"rank (dimension of FFT) = " << 3 << std::endl);
            FIXOUT(config::Log,"How many (FFT's) = " << howmany << std::endl);
            FIXOUTVEC(config::Log,"inembed = ",onembed[0],onembed[1],onembed[2]);
            FIXOUT(config::Log,"istride = " << ostride << std::endl);
            FIXOUT(config::Log,"idist = " << odist << std::endl);
            FIXOUTVEC(config::Log,"onembed = ",inembed[0],inembed[1],inembed[2]);
            FIXOUT(config::Log,"ostride = " << istride << std::endl);
            FIXOUT(config::Log,"odist = " << idist << std::endl);
            FIXOUT(config::Log,"Direction (sign) = " << "HIPFFT_BACKWARD" << std::endl);
            if(hipfftPlanMany(&FPc2c,3,n,onembed,ostride,odist,inembed,istride,idist,HIPFFT_C2C,howmany)!=HIPFFT_SUCCESS)
            {
                error::errPreamble(__FILE__,__LINE__);
                error::errMessage("CUFFT 3D plan creation failed");
            }
            else
            {
                FIXOUT(config::Log,"CUFFT returned success");
            }

            //At this point we can copy the interaction matrix from the CPU
            //as there is no need to do the determination of the interaction
            //matrix on the card.
            //declare a holder on the heap
            if(config::exchm==0)
            {
                Array7D<fftwf_complex> tempNkab;
                tempNkab.resize(geom::ucm.GetNMS(),geom::ucm.GetNMS(),3,3,geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::zpdim[2]*geom::Nk[2]);
                for(unsigned int s1 = 0 ; s1 < geom::ucm.GetNMS() ; s1++)
                {
                    for(unsigned int s2 = 0 ; s2 < geom::ucm.GetNMS() ; s2++)
                    {
                        for(unsigned int alpha = 0 ; alpha < 3 ; alpha++)
                        {
                            for(unsigned int beta = 0 ; beta < 3 ; beta++)
                            {
                                for(unsigned int i = 0 ; i < geom::zpdim[0]*geom::Nk[0] ; i++)
                                {
                                    for(unsigned int j = 0 ; j < geom::zpdim[1]*geom::Nk[1] ; j++)
                                    {
                                        for(unsigned int k = 0 ; k < geom::zpdim[2]*geom::Nk[2] ; k++)
                                        {
                                            for(unsigned int l = 0 ; l < 2 ; l++)
                                            {
                                                tempNkab(s1,s2,alpha,beta,i,j,k)[l]=static_cast<float>(intmat::Nkab(s1,s2,alpha,beta,i,j,k)[l]);
                                            }

                                        }
                                    }
                                }
                            }
                        }
                    }
                }
                //copy the FT'd interaction matrix to the card
                CUDA_CALL(hipMemcpy(CNk,tempNkab.ptr(),geom::ucm.GetNMS()*geom::ucm.GetNMS()*3*3*geom::zpdim[0]*geom::zpdim[1]*geom::zpdim[2]*geom::Nk[0]*geom::Nk[1]*geom::Nk[2]*sizeof(hipfftComplex),hipMemcpyHostToDevice));
                intmat::Nkab.clear();
                //clear the floating point holding arrays as well
                tempNkab.clear();
                check_cuda_errors(__FILE__,__LINE__);

            }
            else if(config::exchm>0 && config::dipm==0 && config::inc_dip==true)
            {
                Array5D<fftwf_complex> tempNkab;
                tempNkab.resize(3,3,geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::zpdim[2]*geom::Nk[2]);
                tempNkab.IFill(0);
                for(unsigned int alpha = 0 ; alpha < 3 ; alpha++)
                {
                    for(unsigned int beta = 0 ; beta < 3 ; beta++)
                    {
                        for(unsigned int i = 0 ; i < geom::zpdim[0]*geom::Nk[0] ; i++)
                        {
                            for(unsigned int j = 0 ; j < geom::zpdim[1]*geom::Nk[1] ; j++)
                            {
                                for(unsigned int k = 0 ; k < geom::zpdim[2]*geom::Nk[2] ; k++)
                                {
                                    for(unsigned int l = 0 ; l < 2 ; l++)
                                    {
                                        tempNkab(alpha,beta,i,j,k)[l]=static_cast<float>(intmat::dipNkab(alpha,beta,i,j,k)[l]);
                                    }
                                }
                            }
                        }
                    }
                }
                //copy the FT'd interaction matrix to the card
                CUDA_CALL(hipMemcpy(CNk,tempNkab.ptr(),3*3*geom::zpdim[0]*geom::zpdim[1]*geom::zpdim[2]*geom::Nk[0]*geom::Nk[1]*geom::Nk[2]*sizeof(hipfftComplex),hipMemcpyHostToDevice));
                //intmat::dipNkab.clear();
                //clear the floating point holding arrays as well
//                tempNkab.clear();
                check_cuda_errors(__FILE__,__LINE__);

            }
        }

    }
    void deallocate_cuda_memory()
    {
        config::printline(config::Info);
        config::Info.width(45);config::Info << std::right << "*" << "**EXIT information***" << std::endl;
        FIXOUT(config::Info,"Freeing space on GPU device" << std::flush);
        CUDA_CALL(hipFree(CNk));
        CUDA_CALL(hipFree(CSr));
        CUDA_CALL(hipFree(CSk));
        CUDA_CALL(hipFree(CHk));
        CUDA_CALL(hipFree(CHr));
        CUDA_CALL(hipFree(Cspin));
        CUDA_CALL(hipFree(Cespin));
        if(llg::intscheme==0)
        {
            CUDA_CALL(hipFree(Cfn));
        }
        else if(llg::intscheme==1)
        {
            CUDA_CALL(hipFree(CRK4k1));
            CUDA_CALL(hipFree(CRK4k2));
            CUDA_CALL(hipFree(CRK4k3));
            CUDA_CALL(hipFree(CRK4k4));
        }
        CUDA_CALL(hipFree(CDetFields));
        CUDA_CALL(hipFree(Crand));
        CUDA_CALL(hipFree(CH));
        CUDA_CALL(hipFree(CHstg));
        CUDA_CALL(hipFree(CInitHstg));
        CUDA_CALL(hipFree(Csigma));
        CUDA_CALL(hipFree(Clambda));
        CUDA_CALL(hipFree(Cllgpf));
        CUDA_CALL(hipFree(Cspec));
        CUDA_CALL(hipFree(Ckx));
        CUDA_CALL(hipFree(Cky));
        CUDA_CALL(hipFree(Ckz));
        CUDA_CALL(hipFree(CHDemag));
        CUDA_CALL(hipFree(Cdxx));
        CUDA_CALL(hipFree(Cdxy));
        CUDA_CALL(hipFree(Cdxz));
        CUDA_CALL(hipFree(Cdyx));
        CUDA_CALL(hipFree(Cdyy));
        CUDA_CALL(hipFree(Cdyz));
        CUDA_CALL(hipFree(Cdzx));
        CUDA_CALL(hipFree(Cdzy));
        CUDA_CALL(hipFree(Cdzz));
        CUDA_CALL(hipFree(Cmagmom));
        CUDA_CALL(hipFree(Cdiagoffset));
        CUDA_CALL(hipFree(Coffdiagoffset));
        config::Info << "Done" << std::endl;
    }
    void spins_forward()
    {
        CUFFT_CALL(hipfftExecC2C(SPc2c,CSr,CSk,HIPFFT_FORWARD));
    }

    void fields_back()
    {
        CUFFT_CALL(hipfftExecC2C(FPc2c,CHk,CHr,HIPFFT_BACKWARD));
    }

    void allocate_memory_on_card()
    {
        //all of the GPU memory allocations should happen here.
        //--------------------------------------------------------------------------------
        if(config::exchm==0)
        {
            CUDA_CALL(hipMalloc((void**)&CNk,geom::ucm.GetNMS()*geom::ucm.GetNMS()*3*3*geom::zps*sizeof(hipfftComplex)));
            CUDA_CALL(hipMalloc((void**)&CSk,geom::ucm.GetNMS()*3*geom::zps*sizeof(hipfftComplex)));
            CUDA_CALL(hipMalloc((void**)&CSr,geom::ucm.GetNMS()*3*geom::zps*sizeof(hipfftComplex)));
            CUDA_CALL(hipMalloc((void**)&CHk,geom::ucm.GetNMS()*3*geom::zps*sizeof(hipfftComplex)));
            CUDA_CALL(hipMalloc((void**)&CHr,geom::ucm.GetNMS()*3*geom::zps*sizeof(hipfftComplex)));
            CUDA_CALL(hipMalloc((void**)&Cspec,geom::nspins*sizeof(unsigned int)));
        }
        else if(config::exchm>0)
        {
            if(config::inc_dip==true && config::dipm==0)
            {
                CUDA_CALL(hipMalloc((void**)&CNk,3*3*geom::zps*sizeof(hipfftComplex)));
                CUDA_CALL(hipMalloc((void**)&CSk,3*geom::zps*sizeof(hipfftComplex)));
                CUDA_CALL(hipMalloc((void**)&CSr,3*geom::zps*sizeof(hipfftComplex)));
                CUDA_CALL(hipMalloc((void**)&CHk,3*geom::zps*sizeof(hipfftComplex)));
                CUDA_CALL(hipMalloc((void**)&CHr,3*geom::zps*sizeof(hipfftComplex)));
                CUDA_CALL(hipMalloc((void**)&Cmagmom,geom::nspins*sizeof(float)));
            }

            if(config::exchm==1)//DIA
            {
                CUDA_CALL(hipMalloc((void**)&Cdiagoffset,exch::diagoffset.size()*sizeof(int)));
                //copy the diag offsets
                CUDA_CALL(hipMemcpy(Cdiagoffset,exch::diagoffset.ptr(),exch::diagoffset.size()*sizeof(int),hipMemcpyHostToDevice));
                CUDA_CALL(hipMalloc((void**)&Cdxx,exch::dataxx.size()*sizeof(double)));
                CUDA_CALL(hipMalloc((void**)&Cdyy,exch::datayy.size()*sizeof(double)));
                CUDA_CALL(hipMalloc((void**)&Cdzz,exch::datazz.size()*sizeof(double)));
                //dataxx, datayy and datazz should all be the same size
                double *temp=new double[exch::dataxx.size()];
                for(unsigned int i = 0 ; i < exch::dataxx.size() ; i++)
                {
                    temp[i]=exch::dataxx[i];
                }
                CUDA_CALL(hipMemcpy(Cdxx,temp,exch::dataxx.size()*sizeof(double),hipMemcpyHostToDevice));
                if(exch::dataxx.size()!=exch::datayy.size())
                {
                    error::errPreamble(__FILE__,__LINE__);
                    error::errMessage("Something has gone wrong, the length of the diagonal (xx,yy and zz) components of the exchange\ntensor in the DIA format (exch::dataxx etc) should be the same. Here xx and yy not equal.");
                }
                for(unsigned int i = 0 ; i < exch::dataxx.size() ; i++)
                {
                    temp[i]=exch::datayy[i];
                }
                CUDA_CALL(hipMemcpy(Cdyy,temp,exch::datayy.size()*sizeof(double),hipMemcpyHostToDevice));
                if(exch::dataxx.size()!=exch::datazz.size())
                {
                    error::errPreamble(__FILE__,__LINE__);
                    error::errMessage("Something has gone wrong, the length of the diagonal (xx,yy and zz) components of the exchange\ntensor in the DIA format (exch::dataxx etc) should be the same. Here xx and yy not equal.");
                }
                for(unsigned int i = 0 ; i < exch::dataxx.size() ; i++)
                {
                    temp[i]=exch::datayy[i];
                }
                CUDA_CALL(hipMemcpy(Cdzz,temp,exch::datazz.size()*sizeof(double),hipMemcpyHostToDevice));
                if(config::offdiag==true)
                {
                    CUDA_CALL(hipMalloc((void**)&Coffdiagoffset,exch::offdiagoffset.size()*sizeof(int)));
                    CUDA_CALL(hipMemcpy(Coffdiagoffset,exch::offdiagoffset.ptr(),exch::offdiagoffset.size()*sizeof(int),hipMemcpyHostToDevice));
                    if(!(exch::dataxy.size()==exch::dataxz.size()==exch::datayx.size()==exch::datayz.size()==exch::datazx.size()==exch::datazy.size()))
                    {
                        error::errPreamble(__FILE__,__LINE__);
                        error::errMessage("Size of the off diagonal exchange components (in DIA format) are not of the same length.");
                    }
                    CUDA_CALL(hipMalloc((void**)&Cdxy,exch::dataxy.size()*sizeof(double)));
                    CUDA_CALL(hipMalloc((void**)&Cdxz,exch::dataxz.size()*sizeof(double)));
                    CUDA_CALL(hipMalloc((void**)&Cdyx,exch::datayx.size()*sizeof(double)));
                    CUDA_CALL(hipMalloc((void**)&Cdyz,exch::datayz.size()*sizeof(double)));
                    CUDA_CALL(hipMalloc((void**)&Cdzx,exch::datazx.size()*sizeof(double)));
                    CUDA_CALL(hipMalloc((void**)&Cdzy,exch::datazy.size()*sizeof(double)));
                    double *otemp=new double[exch::dataxy.size()];for(unsigned int i = 0 ; i < exch::dataxy.size() ; i++){otemp[i]=static_cast<double>(exch::dataxy[i]);}
                    CUDA_CALL(hipMemcpy(Cdxy,otemp,exch::dataxy.size()*sizeof(double),hipMemcpyHostToDevice));
                    for(unsigned int i = 0 ; i < exch::dataxz.size() ; i++){otemp[i]=static_cast<double>(exch::dataxz[i]);}
                    CUDA_CALL(hipMemcpy(Cdxz,otemp,exch::dataxz.size()*sizeof(double),hipMemcpyHostToDevice));
                    for(unsigned int i = 0 ; i < exch::datayx.size() ; i++){otemp[i]=static_cast<double>(exch::datayx[i]);}
                    CUDA_CALL(hipMemcpy(Cdyx,otemp,exch::datayx.size()*sizeof(double),hipMemcpyHostToDevice));
                    for(unsigned int i = 0 ; i < exch::datayz.size() ; i++){otemp[i]=static_cast<double>(exch::datayz[i]);}
                    CUDA_CALL(hipMemcpy(Cdyz,otemp,exch::datayz.size()*sizeof(double),hipMemcpyHostToDevice));
                    for(unsigned int i = 0 ; i < exch::datazx.size() ; i++){otemp[i]=static_cast<double>(exch::datazx[i]);}
                    CUDA_CALL(hipMemcpy(Cdzx,otemp,exch::datazx.size()*sizeof(double),hipMemcpyHostToDevice));
                    for(unsigned int i = 0 ; i < exch::datazy.size() ; i++){otemp[i]=static_cast<double>(exch::datazy[i]);}
                    CUDA_CALL(hipMemcpy(Cdzy,otemp,exch::datazy.size()*sizeof(double),hipMemcpyHostToDevice));
                }
            }
            else if(config::exchm==2)//CSR
            {
                CUDA_CALL(hipMalloc((void**)&Cxadj,exch::xadj.size()*sizeof(unsigned int)));
                CUDA_CALL(hipMalloc((void**)&Cadjncy,exch::adjncy.size()*sizeof(unsigned int)));
                CUDA_CALL(hipMemcpy(Cxadj,exch::xadj.ptr(),exch::xadj.size()*sizeof(unsigned int),hipMemcpyHostToDevice));
                CUDA_CALL(hipMemcpy(Cadjncy,exch::adjncy.ptr(),exch::adjncy.size()*sizeof(unsigned int),hipMemcpyHostToDevice));
                CUDA_CALL(hipMalloc((void**)&Cdxx,exch::dataxx.size()*sizeof(double)));
                CUDA_CALL(hipMalloc((void**)&Cdyy,exch::datayy.size()*sizeof(double)));
                CUDA_CALL(hipMalloc((void**)&Cdzz,exch::datazz.size()*sizeof(double)));
                //dataxx, datayy and datazz should all be the same size
                double *temp=new double[exch::dataxx.size()];
                for(unsigned int i = 0 ; i < exch::dataxx.size() ; i++)
                {
                    temp[i]=static_cast<double>(exch::dataxx[i]);
                }
                CUDA_CALL(hipMemcpy(Cdxx,temp,exch::dataxx.size()*sizeof(double),hipMemcpyHostToDevice));
                if(exch::dataxx.size()!=exch::datayy.size())
                {
                    error::errPreamble(__FILE__,__LINE__);
                    error::errMessage("Something has gone wrong, the length of the diagonal (xx,yy and zz) components of the exchange\ntensor in the CSR format (exch::dataxx etc) should be the same. Here xx and yy not equal.");
                }
                for(unsigned int i = 0 ; i < exch::dataxx.size() ; i++)
                {
                    temp[i]=static_cast<double>(exch::datayy[i]);
                }
                CUDA_CALL(hipMemcpy(Cdyy,temp,exch::datayy.size()*sizeof(double),hipMemcpyHostToDevice));
                if(exch::dataxx.size()!=exch::datazz.size())
                {
                    error::errPreamble(__FILE__,__LINE__);
                    error::errMessage("Something has gone wrong, the length of the diagonal (xx,yy and zz) components of the exchange\ntensor in the CSR format (exch::dataxx etc) should be the same. Here xx and yy not equal.");
                }
                for(unsigned int i = 0 ; i < exch::dataxx.size() ; i++)
                {
                    temp[i]=static_cast<double>(exch::datayy[i]);
                }
                CUDA_CALL(hipMemcpy(Cdzz,temp,exch::datazz.size()*sizeof(double),hipMemcpyHostToDevice));
                if(config::offdiag==true)
                {
                    unsigned int arsize=exch::dataxy.size();
                    if(!(exch::dataxy.size()==arsize && exch::dataxz.size()==arsize && exch::datayx.size()==arsize && exch::datayz.size()==arsize && exch::datazx.size()==arsize && exch::datazy.size()==arsize))
                    {
                        error::errPreamble(__FILE__,__LINE__);
                    //    std::cout << exch::dataxy.size() << "\t"<< exch::dataxz.size() << "\t"<< exch::datayx.size() << "\t"<< exch::datayz.size() << "\t"<< exch::datazx.size() << "\t"<< exch::datazy.size() << std::endl;
                        error::errMessage("Size of the off diagonal exchange components (in DIA format) are not of the same length.");
                    }
                    CUDA_CALL(hipMalloc((void**)&Cdxy,exch::dataxy.size()*sizeof(double)));
                    CUDA_CALL(hipMalloc((void**)&Cdxz,exch::dataxz.size()*sizeof(double)));
                    CUDA_CALL(hipMalloc((void**)&Cdyx,exch::datayx.size()*sizeof(double)));
                    CUDA_CALL(hipMalloc((void**)&Cdyz,exch::datayz.size()*sizeof(double)));
                    CUDA_CALL(hipMalloc((void**)&Cdzx,exch::datazx.size()*sizeof(double)));
                    CUDA_CALL(hipMalloc((void**)&Cdzy,exch::datazy.size()*sizeof(double)));
                    double *otemp=new double[exch::dataxy.size()];for(unsigned int i = 0 ; i < exch::dataxy.size() ; i++){otemp[i]=static_cast<double>(exch::dataxy[i]);}
                    CUDA_CALL(hipMemcpy(Cdxy,otemp,exch::dataxy.size()*sizeof(double),hipMemcpyHostToDevice));
                    for(unsigned int i = 0 ; i < exch::dataxz.size() ; i++){otemp[i]=static_cast<double>(exch::dataxz[i]);}
                    CUDA_CALL(hipMemcpy(Cdxz,otemp,exch::dataxz.size()*sizeof(double),hipMemcpyHostToDevice));
                    for(unsigned int i = 0 ; i < exch::datayx.size() ; i++){otemp[i]=static_cast<double>(exch::datayx[i]);}
                    CUDA_CALL(hipMemcpy(Cdyx,otemp,exch::datayx.size()*sizeof(double),hipMemcpyHostToDevice));
                    for(unsigned int i = 0 ; i < exch::datayz.size() ; i++){otemp[i]=static_cast<double>(exch::datayz[i]);}
                    CUDA_CALL(hipMemcpy(Cdyz,otemp,exch::datayz.size()*sizeof(double),hipMemcpyHostToDevice));
                    for(unsigned int i = 0 ; i < exch::datazx.size() ; i++){otemp[i]=static_cast<double>(exch::datazx[i]);}
                    CUDA_CALL(hipMemcpy(Cdzx,otemp,exch::datazx.size()*sizeof(double),hipMemcpyHostToDevice));
                    for(unsigned int i = 0 ; i < exch::datazy.size() ; i++){otemp[i]=static_cast<double>(exch::datazy[i]);}
                    CUDA_CALL(hipMemcpy(Cdzy,otemp,exch::datazy.size()*sizeof(double),hipMemcpyHostToDevice));
                }
            }
        }

        if(exch::inc4spin)
        {
            CUDA_CALL(hipMalloc((void**)&Cxadj_jkl,exch::xadj_j.size()*sizeof(unsigned int)));
            CUDA_CALL(hipMalloc((void**)&Cadjncy_j,exch::adjncy_j.size()*sizeof(unsigned int)));
            CUDA_CALL(hipMalloc((void**)&Cadjncy_k,exch::adjncy_k.size()*sizeof(unsigned int)));
            CUDA_CALL(hipMalloc((void**)&Cadjncy_l,exch::adjncy_l.size()*sizeof(unsigned int)));
            CUDA_CALL(hipMemcpy(Cxadj_jkl,exch::xadj_j.ptr(),exch::xadj_j.size()*sizeof(unsigned int),hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(Cadjncy_j,exch::adjncy_j.ptr(),exch::adjncy_j.size()*sizeof(unsigned int),hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(Cadjncy_k,exch::adjncy_k.ptr(),exch::adjncy_k.size()*sizeof(unsigned int),hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(Cadjncy_l,exch::adjncy_l.ptr(),exch::adjncy_l.size()*sizeof(unsigned int),hipMemcpyHostToDevice));
        }
        CUDA_CALL(hipMalloc((void**)&CHDemag,3*geom::nspins*sizeof(float)));
        CUDA_CALL(hipMalloc((void**)&CHstg,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&CInitHstg,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Cspin,3*geom::nspins*sizeof(double)));
        if(llg::intscheme==0)
        {
            CUDA_CALL(hipMalloc((void**)&Cespin,3*geom::nspins*sizeof(double)));
            CUDA_CALL(hipMalloc((void**)&Cfn,3*geom::nspins*sizeof(double)));
        }
        else if(llg::intscheme==1)//RK4
        {
            CUDA_CALL(hipMalloc((void**)&CRK4k1,3*geom::nspins*sizeof(double)));
            CUDA_CALL(hipMalloc((void**)&CRK4k2,3*geom::nspins*sizeof(double)));
            CUDA_CALL(hipMalloc((void**)&CRK4k3,3*geom::nspins*sizeof(double)));
            CUDA_CALL(hipMalloc((void**)&CRK4k4,3*geom::nspins*sizeof(double)));
        }

        CUDA_CALL(hipMalloc((void**)&CDetFields,3*geom::nspins*sizeof(double)));
        curandN=3*geom::nspins;
        if(curandN%2!=0)//Check if we have an odd number of random numbers (MUST BE EVEN!!!)
        {
            curandN=curandN+1;
        }
        CUDA_CALL(hipMalloc((void**)&Crand,curandN*sizeof(float)));
        CUDA_CALL(hipMalloc((void**)&Ck1udir,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&CH,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Clambda,geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Csigma,geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Cllgpf,geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Ck1u,geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Ckx,geom::nspins*sizeof(unsigned int)));
        CUDA_CALL(hipMalloc((void**)&Cky,geom::nspins*sizeof(unsigned int)));
        CUDA_CALL(hipMalloc((void**)&Ckz,geom::nspins*sizeof(unsigned int)));
        //--------------------------------------------------------------------------------
        //this section sorts out the copying of the data from the CPU to the card
        //--------------------------------------------------------------------------------
        //copy the sigma prefactor
        CUDA_CALL(hipMemcpy(Csigma,geom::sigma.ptr(),geom::nspins*sizeof(double),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(Clambda,geom::lambda.ptr(),geom::nspins*sizeof(double),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(Cllgpf,geom::llgpf.ptr(),geom::nspins*sizeof(double),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(Ck1u,anis::k1u.ptr(),geom::nspins*sizeof(double),hipMemcpyHostToDevice));
        //declare some arrays for doing copying to card
        //Nspins float array, 3*Nspins float array.
        float *nsfa=new float[geom::nspins];
        float *tnsfa=new float[3*geom::nspins];
        //Nspins double array, 3*Nspins double array
        double *nsda=new double[geom::nspins];
        double *tnsda=new double[3*geom::nspins];
        //Nspins int array, 3*Nspins int array
        int *nsia=new int[geom::nspins];
        int *tnsia=new int[3*geom::nspins];
        //copy the 3 Nspin long arrays containing the staggered field to the tnsfa array
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            tnsda[3*i]=  fields::Hstagx(i);
            tnsda[3*i+1]=fields::Hstagy(i);
            tnsda[3*i+2]=fields::Hstagz(i);
        }
        CUDA_CALL(hipMemcpy(CHstg,tnsda,geom::nspins*3*sizeof(double),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(CInitHstg,tnsda,geom::nspins*3*sizeof(double),hipMemcpyHostToDevice));

        //copy the first order uniaxial anisotropy direction
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            for(unsigned int coord = 0 ; coord < 3 ; coord++)
            {
                tnsda[3*i+coord]=anis::k1udir(i,coord);
            }
        }
        CUDA_CALL(hipMemcpy(Ck1udir,tnsda,geom::nspins*3*sizeof(double),hipMemcpyHostToDevice));
        //copy the location of the spins in real space to the device
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            nsia[i]=geom::lu(i,0);
        }
        CUDA_CALL(hipMemcpy(Ckx,nsia,geom::nspins*sizeof(unsigned int),hipMemcpyHostToDevice));
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            nsia[i]=geom::lu(i,1);
        }
        CUDA_CALL(hipMemcpy(Cky,nsia,geom::nspins*sizeof(unsigned int),hipMemcpyHostToDevice));
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            nsia[i]=geom::lu(i,2);
        }
        CUDA_CALL(hipMemcpy(Ckz,nsia,geom::nspins*sizeof(unsigned int),hipMemcpyHostToDevice));
        util::copy3vecto1(geom::nspins,spins::Sx,spins::Sy,spins::Sz,tnsda);
        //copy spin data to single array
        //copy spin data to card
        FIXOUT(config::Log,"Copying spin memory to device:" << std::flush);
        CUDA_CALL(hipMemcpy(Cspin,tnsda,3*geom::nspins*sizeof(double),hipMemcpyHostToDevice));
        SUCCESS(config::Log);
        //zero the field array
        for(unsigned int i = 0 ; i < 3*geom::nspins ; i++){tnsfa[i]=0.0;}CUDA_CALL(hipMemcpy(CH,tnsfa,3*geom::nspins*sizeof(float),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(CHDemag,tnsfa,3*geom::nspins*sizeof(float),hipMemcpyHostToDevice));

        if(config::exchm==0)
        {
            //and copy the species list
            for(unsigned int i = 0 ; i < geom::nspins ; i++)
            {
                nsia[i]=geom::lu(i,3);
            }
            CUDA_CALL(hipMemcpy(Cspec,nsia,geom::nspins*sizeof(unsigned int),hipMemcpyHostToDevice));
        }
        else if(config::exchm>0 && config::dipm==0 && config::inc_dip==true)
        {
            //and copy the species list
            for(unsigned int i = 0 ; i < geom::nspins ; i++)
            {
                nsfa[i]=geom::mu[i];
            }
            CUDA_CALL(hipMemcpy(Cmagmom,nsfa,geom::nspins*sizeof(float),hipMemcpyHostToDevice));
        }


        //make sure we clean up when the program exits
        atexit(deallocate_cuda_memory);
    }
    void CsetStagFields()
    {
        double *tnsda=new double[3*geom::nspins];
        //copy the 3 Nspin long arrays containing the staggered field to the tnsfa array
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            tnsda[3*i]=  fields::Hstagx(i);
            tnsda[3*i+1]=fields::Hstagy(i);
            tnsda[3*i+2]=fields::Hstagz(i);
        }
        CUDA_CALL(hipMemcpy(CHstg,tnsda,geom::nspins*3*sizeof(double),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(CInitHstg,tnsda,geom::nspins*3*sizeof(double),hipMemcpyHostToDevice));
    }
    void CsetStagFieldsZero()
    {
        double *tnsda=new double[3*geom::nspins];
        //copy the 3 Nspin long arrays containing the staggered field to the tnsfa array
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            tnsda[3*i]=  0.0;
            tnsda[3*i+1]=0.0;
            tnsda[3*i+2]=0.0;
        }
        CUDA_CALL(hipMemcpy(CHstg,tnsda,geom::nspins*3*sizeof(double),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(CInitHstg,tnsda,geom::nspins*3*sizeof(double),hipMemcpyHostToDevice));
    }
}
