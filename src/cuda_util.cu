// File: cuda.cu
// Author:Tom Ostler
// Created: 26/06/2014
// Last-modified: 23 Sep 2014 12:24:35
#include "../inc/hip/hip_runtime.h"
#include "../inc/config.h"
#include "../inc/spins.h"
#include "../inc/mat.h"
#include "../inc/geom.h"
#include "../inc/config.h"
#include "../inc/random.h"
#include "../inc/intmat.h"
#include "../inc/util.h"
#include "../inc/fields.h"
#include "../inc/arrays.h"
#include "../inc/cudadefs.h"
#include "../inc/defines.h"
#include "../inc/cufields.h"
#include "../inc/cuint.h"
#include "../inc/llg.h"
//Cuda headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
//Library headers
#include <cmath>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <iostream>
//The function of this file is to house a number of routines
//that deal with a number of underlying routines, such as
//mallocing/de(m)allocing memory, setting up fft's etc.
// Requires: cullg::cuinit() to be called
namespace cullg
{

    void setup_fourier_transform()
    {
        //Even though we have 9 interaction matrices, 3 field arrays and
        //3 spin arrays we only need one transform in cufft. This is because
        //we can reuse the plan and alternate the sign depending on whether
        //we have a forward or a back transform
        /*Create a 3D FFT plan. */
        if(hipfftPlan3d(&C3DPr2c,geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::zpdim[2]*geom::Nk[2],HIPFFT_R2C)!=HIPFFT_SUCCESS)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("CUFFT 3D plan creation failed");
        }
        if(hipfftPlan3d(&C3DPc2r,geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::zpdim[2]*geom::Nk[2],HIPFFT_C2R)!=HIPFFT_SUCCESS)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("CUFFT 3D plan creation failed");
        }


        //At this point we can copy the interaction matrix from the CPU
        //as there is no need to do the determination of the interaction
        //matrix on the card.
        //declare a holder on the heap
        Array3D<fftwf_complex> tempxx,tempxy,tempxz,tempyx,tempyy,tempyz,tempzx,tempzy,tempzz;
        tempxx.resize(geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::cplxdim);
        tempxy.resize(geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::cplxdim);
        tempxz.resize(geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::cplxdim);
        tempyx.resize(geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::cplxdim);
        tempyy.resize(geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::cplxdim);
        tempyz.resize(geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::cplxdim);
        tempzx.resize(geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::cplxdim);
        tempzy.resize(geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::cplxdim);
        tempzz.resize(geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::cplxdim);
        for(unsigned int i = 0 ; i < geom::zpdim[0]*geom::Nk[0] ; i++)
        {
            for(unsigned int j = 0 ; j < geom::zpdim[1]*geom::Nk[1] ; j++)
            {
                for(unsigned int k = 0 ; k < geom::cplxdim ; k++)
                {
                    for(unsigned int l = 0 ; l < 2 ; l++)
                    {
                        tempxx(i,j,k)[l]=float(intmat::Nxx(i,j,k)[l]);
                        tempxy(i,j,k)[l]=float(intmat::Nxy(i,j,k)[l]);
                        tempxz(i,j,k)[l]=float(intmat::Nxz(i,j,k)[l]);
                        tempyx(i,j,k)[l]=float(intmat::Nyx(i,j,k)[l]);
                        tempyy(i,j,k)[l]=float(intmat::Nyy(i,j,k)[l]);
                        tempyz(i,j,k)[l]=float(intmat::Nyz(i,j,k)[l]);
                        tempzx(i,j,k)[l]=float(intmat::Nzx(i,j,k)[l]);
                        tempzy(i,j,k)[l]=float(intmat::Nzy(i,j,k)[l]);
                        tempzz(i,j,k)[l]=float(intmat::Nzz(i,j,k)[l]);
                    }
                }
            }
        }

        //copy the FT'd interaction matrix to the card
        CUDA_CALL(hipMemcpy(CCNxx,tempxx.ptr(),geom::czps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(CCNxy,tempxy.ptr(),geom::czps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(CCNxz,tempxz.ptr(),geom::czps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(CCNyx,tempyx.ptr(),geom::czps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(CCNyy,tempyy.ptr(),geom::czps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(CCNyz,tempyz.ptr(),geom::czps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(CCNzx,tempzx.ptr(),geom::czps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(CCNzy,tempzy.ptr(),geom::czps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(CCNzz,tempzz.ptr(),geom::czps*sizeof(hipfftComplex),hipMemcpyHostToDevice));
        //clear the memory on the CPU
        intmat::Nxx.clear();
        intmat::Nxy.clear();
        intmat::Nxz.clear();
        intmat::Nyx.clear();
        intmat::Nyy.clear();
        intmat::Nyz.clear();
        intmat::Nzx.clear();
        intmat::Nzy.clear();
        intmat::Nzz.clear();
        //clear the floating point holding arrays as well
        tempxx.clear();
        tempxy.clear();
        tempxz.clear();
        tempyx.clear();
        tempyy.clear();
        tempyz.clear();
        tempzx.clear();
        tempzy.clear();
        tempzz.clear();
        check_cuda_errors(__FILE__,__LINE__);
    }
    void deallocate_cuda_memory()
    {
        config::printline(config::Info);
        config::Info.width(45);config::Info << std::right << "*" << "**EXIT information***" << std::endl;
        FIXOUT(config::Info,"Freeing space on GPU device" << std::flush);
        CUDA_CALL(hipFree(CCNxx));
        CUDA_CALL(hipFree(CCNxy));
        CUDA_CALL(hipFree(CCNxz));
        CUDA_CALL(hipFree(CCNyx));
        CUDA_CALL(hipFree(CCNyy));
        CUDA_CALL(hipFree(CCNyz));
        CUDA_CALL(hipFree(CCNzx));
        CUDA_CALL(hipFree(CCNzy));
        CUDA_CALL(hipFree(CCNzz));
        CUDA_CALL(hipFree(CCSrx));
        CUDA_CALL(hipFree(CCSry));
        CUDA_CALL(hipFree(CCSrz));
        CUDA_CALL(hipFree(CCSkx));
        CUDA_CALL(hipFree(CCSky));
        CUDA_CALL(hipFree(CCSkz));
        CUDA_CALL(hipFree(CCHkx));
        CUDA_CALL(hipFree(CCHky));
        CUDA_CALL(hipFree(CCHkz));
        CUDA_CALL(hipFree(CCHrx));
        CUDA_CALL(hipFree(CCHry));
        CUDA_CALL(hipFree(CCHrz));
        CUDA_CALL(hipFree(Cspin));
        CUDA_CALL(hipFree(Cespin));
        CUDA_CALL(hipFree(Crand));
        CUDA_CALL(hipFree(CH));
        CUDA_CALL(hipFree(Czpsn));
        CUDA_CALL(hipFree(Cfn));
        config::Info << "Done" << std::endl;
    }
    void spins_forward()
    {
        CUFFT_CALL(hipfftExecR2C(C3DPr2c,CCSrx,CCSkx));
        CUFFT_CALL(hipfftExecR2C(C3DPr2c,CCSry,CCSky));
        CUFFT_CALL(hipfftExecR2C(C3DPr2c,CCSrz,CCSkz));
    }

    void fields_back()
    {
        CUFFT_CALL(hipfftExecC2R(C3DPc2r,CCHkx,CCHrx));
        CUFFT_CALL(hipfftExecC2R(C3DPc2r,CCHky,CCHry));
        CUFFT_CALL(hipfftExecC2R(C3DPc2r,CCHkz,CCHrz));
    }

    void allocate_memory_on_card()
    {
        //all of the GPU memory allocations should happen here.
        //--------------------------------------------------------------------------------
        CUDA_CALL(hipMalloc((void**)&CCNxx,geom::czps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNxy,geom::czps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNxz,geom::czps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNyx,geom::czps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNyy,geom::czps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNyz,geom::czps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNzx,geom::czps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNzy,geom::czps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCNzz,geom::czps*sizeof(hipfftComplex)));

        CUDA_CALL(hipMalloc((void**)&CCSkx,geom::czps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCSky,geom::czps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCSkz,geom::czps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCSrx,geom::zps*sizeof(hipfftReal)));
        CUDA_CALL(hipMalloc((void**)&CCSry,geom::zps*sizeof(hipfftReal)));
        CUDA_CALL(hipMalloc((void**)&CCSrz,geom::zps*sizeof(hipfftReal)));
        CUDA_CALL(hipMalloc((void**)&CCHkx,geom::czps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCHky,geom::czps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCHkz,geom::czps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CCHrx,geom::zps*sizeof(hipfftReal)));
        CUDA_CALL(hipMalloc((void**)&CCHry,geom::zps*sizeof(hipfftReal)));
        CUDA_CALL(hipMalloc((void**)&CCHrz,geom::zps*sizeof(hipfftReal)));
        CUDA_CALL(hipMalloc((void**)&Cspin,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Cespin,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Crand,3*geom::nspins*sizeof(float)));
        CUDA_CALL(hipMalloc((void**)&CH,3*geom::nspins*sizeof(float)));
        CUDA_CALL(hipMalloc((void**)&Clu,geom::zps*sizeof(int)));
        CUDA_CALL(hipMalloc((void**)&Czpsn,geom::nspins*sizeof(int)));
        CUDA_CALL(hipMalloc((void**)&Cspec,geom::nspins*sizeof(unsigned int)));

        CUDA_CALL(hipMalloc((void**)&Cfn,3*geom::nspins*sizeof(double)));

        //--------------------------------------------------------------------------------
        //this section sorts out the copying of the data from the CPU to the card
        //--------------------------------------------------------------------------------
        //declare some arrays for doing copying to card
        //Nspins float array, 3*Nspins float array.
        float *nsfa=new float[geom::nspins];
        float *tnsfa=new float[3*geom::nspins];
        //Nspins double array, 3*Nspins double array
        double *nsda=new double[geom::nspins];
        double *tnsda=new double[3*geom::nspins];
        //Nspins int array, 3*Nspins int array
        int *nsia=new int[geom::nspins];
        int *tnsia=new int[3*geom::nspins];
        //copy spin data to single array
        util::copy3vecto1(geom::nspins,spins::Sx,spins::Sy,spins::Sz,tnsda);
        //copy spin data to card
        CUDA_CALL(hipMemcpy(Cspin,tnsda,3*geom::nspins*sizeof(double),hipMemcpyHostToDevice));
        //copy the species information to the card
        CUDA_CALL(hipMemcpy(Cspec,geom::spec,geom::nspins*sizeof(unsigned int),hipMemcpyHostToDevice));
        int *sn=NULL;
        sn=new int[geom::zps];
        int count=0;
        for(unsigned int i = 0 ; i < geom::zpdim[0]*geom::Nk[0] ; i++)
        {
            for(unsigned int j = 0 ; j < geom::zpdim[1]*geom::Nk[1] ; j++)
            {
                for(unsigned int k = 0 ; k < geom::zpdim[2]*geom::Nk[2] ; k++)
                {
                    if(geom::coords(i,j,k,0) > -1)
                    {
                        sn[count]=geom::coords(i,j,k,0);
                    }
                    else
                    {
                        sn[count]=-1;
                    }
                    count++;
                }
            }
        }
        if(count!=geom::zps)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errWarning("Error in counting zero pad size for GPU lookup");
        }
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            nsia[i]=spins::Srx.getarrayelement(geom::lu(i,0),geom::lu(i,1),geom::lu(i,2));
            //			std::cerr << nsia[i] << std::endl;
        }
        CUDA_CALL(hipMemcpy(Czpsn,nsia,geom::nspins*sizeof(int),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(Clu,sn,geom::zps*sizeof(int),hipMemcpyHostToDevice));
        //zero the field array
        for(unsigned int i = 0 ; i < 3*geom::nspins ; i++){tnsfa[i]=0.0;}CUDA_CALL(hipMemcpy(CH,tnsfa,3*geom::nspins*sizeof(float),hipMemcpyHostToDevice));
        //		hipfftReal
        //--------------------------------------------------------------------------------


        //make sure we clean up when the program exits
        atexit(deallocate_cuda_memory);
    }
}
