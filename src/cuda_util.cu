#include "hip/hip_runtime.h"
// File: cuda.cu
// Author:Tom Ostler
// Created: 26/06/2014
// Last-modified: 03 Oct 2014 14:43:49
#include "../inc/hip/hip_runtime.h"
#include "../inc/config.h"
#include "../inc/spins.h"
#include "../inc/geom.h"
#include "../inc/config.h"
#include "../inc/random.h"
#include "../inc/intmat.h"
#include "../inc/util.h"
#include "../inc/fields.h"
#include "../inc/arrays.h"
#include "../inc/cudadefs.h"
#include "../inc/defines.h"
#include "../inc/cufields.h"
#include "../inc/cuint.h"
#include "../inc/llg.h"
//Cuda headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
//Library headers
#include <cmath>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <iostream>
//The function of this file is to house a number of routines
//that deal with a number of underlying routines, such as
//mallocing/de(m)allocing memory, setting up fft's etc.
// Requires: cullg::cuinit() to be called
namespace cullg
{

    void setup_fourier_transform()
    {
        /*Create a 3D FFT plan. */
        int n[3]={geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::zpdim[2]*geom::Nk[2]};
        int *inembed=n;
        int *onembed=n;
        int istride=1;
        int ostride=1;
        int idist=geom::zps;
        int odist=geom::zps;
        config::openLogFile();
        config::printline(config::Log);
        FIXOUT(config::Log,"Parameters entering into CUFFT plan of the spin arrays (forward)" << std::endl);
        FIXOUTVEC(config::Log,"Dimensions of FFT = ",n[0],n[1],n[2]);
        FIXOUT(config::Log,"rank (dimension of FFT) = " << 3 << std::endl);
        FIXOUT(config::Log,"How many (FFT's) = " << geom::ucm.GetNMS()*3 << std::endl);
        FIXOUTVEC(config::Log,"inembed = ",inembed[0],inembed[1],inembed[2]);
        FIXOUT(config::Log,"istride = " << istride << std::endl);
        FIXOUT(config::Log,"idist = " << idist << std::endl);
        FIXOUTVEC(config::Log,"onembed = ",onembed[0],onembed[1],onembed[2]);
        FIXOUT(config::Log,"ostride = " << ostride << std::endl);
        FIXOUT(config::Log,"odist = " << odist << std::endl);
        FIXOUT(config::Log,"Direction (sign) = " << "CUFFTW_FORWARD" << std::endl);
        if(hipfftPlanMany(&SPc2c,3,n,inembed,istride,idist,onembed,ostride,odist,HIPFFT_C2C,geom::ucm.GetNMS()*3)!=HIPFFT_SUCCESS)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("CUFFT 3D plan creation failed");
        }
        if(hipfftPlanMany(&FPc2c,3,n,onembed,ostride,odist,inembed,istride,idist,HIPFFT_C2C,geom::ucm.GetNMS()*3)!=HIPFFT_SUCCESS)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("CUFFT 3D plan creation failed");
        }


        //At this point we can copy the interaction matrix from the CPU
        //as there is no need to do the determination of the interaction
        //matrix on the card.
        //declare a holder on the heap
        Array7D<fftwf_complex> tempNkab;
        tempNkab.resize(geom::ucm.GetNMS(),geom::ucm.GetNMS(),3,3,geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::zpdim[2]*geom::Nk[2]);
        for(unsigned int s1 = 0 ; s1 < geom::ucm.GetNMS() ; s1++)
        {
            for(unsigned int s2 = 0 ; s2 < geom::ucm.GetNMS() ; s2++)
            {
                for(unsigned int alpha = 0 ; alpha < 3 ; alpha++)
                {
                    for(unsigned int beta = 0 ; beta < 3 ; beta++)
                    {
                        for(unsigned int i = 0 ; i < geom::zpdim[0]*geom::Nk[0] ; i++)
                        {
                            for(unsigned int j = 0 ; j < geom::zpdim[1]*geom::Nk[1] ; j++)
                            {
                                for(unsigned int k = 0 ; k < geom::zpdim[1]*geom::Nk[2] ; k++)
                                {
                                    for(unsigned int l = 0 ; l < 2 ; l++)
                                    {
                                        tempNkab(s1,s2,alpha,beta,i,j,k)[l]=static_cast<float>(intmat::Nkab(s1,s2,alpha,beta,i,j,k)[l]);
                                    }

                                }
                            }
                        }
                    }
                }
            }
        }

        //copy the FT'd interaction matrix to the card
        CUDA_CALL(hipMemcpy(CNk,tempNkab.ptr(),geom::ucm.GetNMS()*geom::ucm.GetNMS()*3*3*geom::zpdim[0]*geom::zpdim[1]*geom::zpdim[2]*geom::Nk[0]*geom::Nk[1]*geom::Nk[2]*sizeof(fftwf_complex),hipMemcpyHostToDevice));
//        intmat::Nkab.clear();
        //clear the floating point holding arrays as well
        tempNkab.clear();
        check_cuda_errors(__FILE__,__LINE__);
    }
    void deallocate_cuda_memory()
    {
        config::printline(config::Info);
        config::Info.width(45);config::Info << std::right << "*" << "**EXIT information***" << std::endl;
        FIXOUT(config::Info,"Freeing space on GPU device" << std::flush);
        CUDA_CALL(hipFree(CNk));
        CUDA_CALL(hipFree(CSr));
        CUDA_CALL(hipFree(CSk));
        CUDA_CALL(hipFree(CHk));
        CUDA_CALL(hipFree(CHr));
        CUDA_CALL(hipFree(Cspin));
        CUDA_CALL(hipFree(Cespin));
        CUDA_CALL(hipFree(Crand));
        CUDA_CALL(hipFree(CH));
        CUDA_CALL(hipFree(Cfn));
        CUDA_CALL(hipFree(Csigma));
        CUDA_CALL(hipFree(Clambda));
        CUDA_CALL(hipFree(Cllgpf));
        CUDA_CALL(hipFree(Cspec));
        CUDA_CALL(hipFree(Ckx));
        CUDA_CALL(hipFree(Cky));
        CUDA_CALL(hipFree(Ckz));
        config::Info << "Done" << std::endl;
    }
    void spins_forward()
    {
        Array5D<fftwf_complex> temp,temp1;
        temp.resize(geom::ucm.GetNMS(),3,geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::zpdim[2]*geom::Nk[2]);
        temp.IFill(0);
        temp1.resize(geom::ucm.GetNMS(),3,geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::zpdim[2]*geom::Nk[2]);
        temp1.IFill(0);
        CUDA_CALL(hipMemcpy(temp.ptr(),CSr,geom::ucm.GetNMS()*3*geom::zps*sizeof(hipfftComplex),hipMemcpyDeviceToHost));
        for(unsigned int i = 0 ; i < geom::ucm.GetNMS() ; i++)
        {
            for(unsigned int j = 0 ; j < 3 ; j++)
            {
                for(unsigned int k = 0 ; k < geom::zpdim[0]*geom::Nk[0] ; k++)
                {
                    for(unsigned int l = 0 ; l < geom::zpdim[1]*geom::Nk[1] ; l++)
                    {
                        for(unsigned int m = 0 ; m < geom::zpdim[2]*geom::Nk[2] ; m++)
                        {
                            //std::cout << "rs-spins on CPU:\t" << temp(i,j,k,l,m)[0] << "\t" << temp(i,j,k,l,m)[1] << std::endl;

                        }
                    }
                }
            }
        }
        int n[3]={geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::zpdim[2]*geom::Nk[2]};
        int *inembed=n;
        int *onembed=n;
        int istride=1;
        int ostride=1;
        int odist=geom::zps;
        int idist=geom::zps;
        fftwf_plan plan=fftwf_plan_many_dft(3,n,geom::ucm.GetNMS()*3,temp.ptr(),inembed,istride,idist,temp.ptr(),onembed,ostride,odist,FFTW_FORWARD,FFTW_PATIENT);
        fftwf_execute(plan);
        CUFFT_CALL(hipfftExecC2C(SPc2c,CSr,CSk,HIPFFT_FORWARD));
        CUDA_CALL(hipMemcpy(temp1.ptr(),CSk,geom::ucm.GetNMS()*3*geom::zps*sizeof(hipfftComplex),hipMemcpyDeviceToHost));
        for(unsigned int i = 0 ; i < geom::ucm.GetNMS() ; i++)
        {
            for(unsigned int j = 0 ; j < 3 ; j++)
            {
                for(unsigned int k = 0 ; k < geom::zpdim[0]*geom::Nk[0] ; k++)
                {
                    for(unsigned int l = 0 ; l < geom::zpdim[1]*geom::Nk[1] ; l++)
                    {
                        for(unsigned int m = 0 ; m < geom::zpdim[2]*geom::Nk[2] ; m++)
                        {
                            if(fabs(temp(i,j,k,l,m)[0]-temp1(i,j,k,l,m)[0])>1e-4 || fabs(temp(i,j,k,l,m)[1]-temp1(i,j,k,l,m)[1])>1e-4)
                            {
                            //std::cout << "diff on CPU:\t" << temp(i,j,k,l,m)[0]-temp1(i,j,k,l,m)[0] << "\t" << temp(i,j,k,l,m)[1]-temp1(i,j,k,l,m)[1] << std::endl;
                            }

                        }
                    }
                }
            }
        }
        Array7D<fftwf_complex> temp2;
        temp2.resize(geom::ucm.GetNMS(),geom::ucm.GetNMS(),3,3,geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::zpdim[2]*geom::Nk[2]);
        CUDA_CALL(hipMemcpy(temp2.ptr(),CNk,geom::ucm.GetNMS()*geom::ucm.GetNMS()*3*3*geom::zps*sizeof(hipfftComplex),hipMemcpyDeviceToHost));
        temp.IFill(0);
        register unsigned int i = 0,j = 0, k = 0, s1 = 0, s2=0, alpha=0, beta=0;
        for(i = 0 ; i < geom::zpdim[0]*geom::Nk[0] ; i++)
        {
            for(j = 0 ; j < geom::zpdim[1]*geom::Nk[1] ; j++)
            {
                for(k = 0 ; k < geom::zpdim[2]*geom::Nk[2] ; k++)
                {
                    for(s1 = 0 ; s1 < geom::ucm.GetNMS() ; s1++)
                    {
                        for(s2 = 0 ; s2 < geom::ucm.GetNMS() ; s2++)
                        {
                            for(alpha = 0 ; alpha < 3 ; alpha++)
                            {
                                for(beta = 0 ; beta < 3 ; beta++)
                                {
                                    temp(s1,alpha,i,j,k)[0]+=(temp2(s1,s2,alpha,beta,i,j,k)[0]*temp1(s2,beta,i,j,k)[0]-temp2(s1,s2,alpha,beta,i,j,k)[1]*temp1(s2,beta,i,j,k)[1]);
                                    temp(s1,alpha,i,j,k)[1]+=(temp2(s1,s2,alpha,beta,i,j,k)[0]*temp1(s2,beta,i,j,k)[1]+temp2(s1,s2,alpha,beta,i,j,k)[1]*temp1(s2,beta,i,j,k)[0]);
                                    if(fabs(temp2(s1,s2,alpha,beta,i,j,k)[0]-intmat::Nkab(s1,s2,alpha,beta,i,j,k)[0])>0.1 || fabs(temp2(s1,s2,alpha,beta,i,j,k)[1]-intmat::Nkab(s1,s2,alpha,beta,i,j,k)[1])>0.1)
                                    {
                                    std::cout << "GPU/CPU diff\t" << temp2(s1,s2,alpha,beta,i,j,k)[0]-intmat::Nkab(s1,s2,alpha,beta,i,j,k)[0] << "\t" << temp2(s1,s2,alpha,beta,i,j,k)[1]-intmat::Nkab(s1,s2,alpha,beta,i,j,k)[1] <<std::endl;
                                    }
                                }
                            }
                        }
                    }

                }
            }
        }
        for(i = 0 ; i < geom::zpdim[0]*geom::Nk[0] ; i++)
        {
            for(j = 0 ; j < geom::zpdim[1]*geom::Nk[1] ; j++)
            {
                for(k = 0 ; k < geom::zpdim[2]*geom::Nk[2] ; k++)
                {
                    for(s1 = 0 ; s1 < geom::ucm.GetNMS() ; s1++)
                    {
                        for(s2 = 0 ; s2 < geom::ucm.GetNMS() ; s2++)
                        {
                            for(alpha = 0 ; alpha < 3 ; alpha++)
                            {
                                for(beta = 0 ; beta < 3 ; beta++)
                                {
                                    std::cout << temp(s1,alpha,i,j,k)[0] << "\t" << temp(s1,alpha,i,j,k)[1] << std::endl;
                                }
                            }
                        }
                    }

                }
            }
        }
        fftwf_plan plan1=fftwf_plan_many_dft(3,n,geom::ucm.GetNMS()*3,temp.ptr(),inembed,istride,idist,temp.ptr(),onembed,ostride,odist,FFTW_BACKWARD,FFTW_PATIENT);
        fftwf_execute(plan1);
        for(unsigned int i = 0 ; i < geom::ucm.GetNMS() ; i++)
        {
            for(unsigned int j = 0 ; j < 3 ; j++)
            {
                for(unsigned int k = 0 ; k < geom::zpdim[0]*geom::Nk[0] ; k++)
                {
                    for(unsigned int l = 0 ; l < geom::zpdim[1]*geom::Nk[1] ; l++)
                    {
                        for(unsigned int m = 0 ; m < geom::zpdim[2]*geom::Nk[2] ; m++)
                        {
                            std::cout << "k-fields on CPU:\t" << temp(i,j,k,l,m)[0] << "\t" << temp(i,j,k,l,m)[1] << std::endl;

                        }
                    }
                }
            }
        }


    }

    void fields_back()
    {
        Array5D<fftwf_complex> temp;
        temp.resize(geom::ucm.GetNMS(),3,geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::zpdim[2]*geom::Nk[2]);
        temp.IFill(0);
        CUDA_CALL(hipMemcpy(temp.ptr(),CHk,geom::ucm.GetNMS()*3*geom::zps*sizeof(hipfftComplex),hipMemcpyDeviceToHost));
        for(unsigned int i = 0 ; i < geom::ucm.GetNMS() ; i++)
        {
            for(unsigned int j = 0 ; j < 3 ; j++)
            {
                for(unsigned int k = 0 ; k < geom::zpdim[0]*geom::Nk[0] ; k++)
                {
                    for(unsigned int l = 0 ; l < geom::zpdim[1]*geom::Nk[1] ; l++)
                    {
                        for(unsigned int m = 0 ; m < geom::zpdim[2]*geom::Nk[2] ; m++)
                        {
                            std::cout << "k-fields on CPU:\t" << temp(i,j,k,l,m)[0] << "\t" << temp(i,j,k,l,m)[1] << std::endl;

                        }
                    }
                }
            }
        }
        CUFFT_CALL(hipfftExecC2C(FPc2c,CHk,CHr,HIPFFT_BACKWARD));
    }

    void allocate_memory_on_card()
    {
        //all of the GPU memory allocations should happen here.
        //--------------------------------------------------------------------------------
        CUDA_CALL(hipMalloc((void**)&CNk,geom::ucm.GetNMS()*geom::ucm.GetNMS()*3*3*geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CSk,geom::ucm.GetNMS()*3*geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CSr,geom::ucm.GetNMS()*3*geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CHk,geom::ucm.GetNMS()*3*geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CHr,geom::ucm.GetNMS()*3*geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&Cspin,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Cespin,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Crand,3*geom::nspins*sizeof(float)));
        CUDA_CALL(hipMalloc((void**)&CH,3*geom::nspins*sizeof(float)));
        CUDA_CALL(hipMalloc((void**)&Cfn,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Clambda,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Csigma,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Cllgpf,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Ckx,geom::nspins*sizeof(unsigned int)));
        CUDA_CALL(hipMalloc((void**)&Cky,geom::nspins*sizeof(unsigned int)));
        CUDA_CALL(hipMalloc((void**)&Ckz,geom::nspins*sizeof(unsigned int)));
        CUDA_CALL(hipMalloc((void**)&Cspec,geom::nspins*sizeof(unsigned int)));
        //--------------------------------------------------------------------------------
        //this section sorts out the copying of the data from the CPU to the card
        //--------------------------------------------------------------------------------
        //copy the sigma prefactor
        CUDA_CALL(hipMemcpy(Csigma,geom::sigma.ptr(),geom::nspins*sizeof(double),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(Clambda,geom::lambda.ptr(),geom::nspins*sizeof(double),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(Cllgpf,geom::llgpf.ptr(),geom::nspins*sizeof(double),hipMemcpyHostToDevice));
        //declare some arrays for doing copying to card
        //Nspins float array, 3*Nspins float array.
        float *nsfa=new float[geom::nspins];
        float *tnsfa=new float[3*geom::nspins];
        //Nspins double array, 3*Nspins double array
        double *nsda=new double[geom::nspins];
        double *tnsda=new double[3*geom::nspins];
        //Nspins int array, 3*Nspins int array
        int *nsia=new int[geom::nspins];
        //copy the location of the spins in real space to the device
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            nsia[i]=geom::lu(i,0);
        }
        CUDA_CALL(hipMemcpy(Ckx,nsia,geom::nspins*sizeof(unsigned int),hipMemcpyHostToDevice));
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            nsia[i]=geom::lu(i,1);
        }
        CUDA_CALL(hipMemcpy(Cky,nsia,geom::nspins*sizeof(unsigned int),hipMemcpyHostToDevice));
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            nsia[i]=geom::lu(i,2);
        }
        CUDA_CALL(hipMemcpy(Ckz,nsia,geom::nspins*sizeof(unsigned int),hipMemcpyHostToDevice));
        //and copy the species list
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            nsia[i]=geom::lu(i,3);
        }
        CUDA_CALL(hipMemcpy(Cspec,nsia,geom::nspins*sizeof(unsigned int),hipMemcpyHostToDevice));
        int *tnsia=new int[3*geom::nspins];
        //copy spin data to single array
        util::copy3vecto1(geom::nspins,spins::Sx,spins::Sy,spins::Sz,tnsda);
        //copy spin data to card
        CUDA_CALL(hipMemcpy(Cspin,tnsda,3*geom::nspins*sizeof(double),hipMemcpyHostToDevice));
        //zero the field array
        for(unsigned int i = 0 ; i < 3*geom::nspins ; i++){tnsfa[i]=0.0;}CUDA_CALL(hipMemcpy(CH,tnsfa,3*geom::nspins*sizeof(float),hipMemcpyHostToDevice));
        //call the kernel to zero the spin array


        //make sure we clean up when the program exits
        atexit(deallocate_cuda_memory);
    }
}
