// File: cuda.cu
// Author:Tom Ostler
// Created: 26/06/2014
// Last-modified: 02 Oct 2014 10:50:31
#include "../inc/hip/hip_runtime.h"
#include "../inc/config.h"
#include "../inc/spins.h"
#include "../inc/geom.h"
#include "../inc/config.h"
#include "../inc/random.h"
#include "../inc/intmat.h"
#include "../inc/util.h"
#include "../inc/fields.h"
#include "../inc/arrays.h"
#include "../inc/cudadefs.h"
#include "../inc/defines.h"
#include "../inc/cufields.h"
#include "../inc/cuint.h"
#include "../inc/llg.h"
//Cuda headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
//Library headers
#include <cmath>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <iostream>
//The function of this file is to house a number of routines
//that deal with a number of underlying routines, such as
//mallocing/de(m)allocing memory, setting up fft's etc.
// Requires: cullg::cuinit() to be called
namespace cullg
{

    void setup_fourier_transform()
    {
        /*Create a 3D FFT plan. */
        int n[3]={geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::zpdim[2]*geom::Nk[2]};
        int *inembed=n;
        int *onembed=n;
        int istride=1;
        int ostride=1;
        int idist=geom::zps;
        int odist=geom::zps;
        config::openLogFile();
        config::printline(config::Log);
        FIXOUT(config::Log,"Parameters entering into CUFFT plan of the spin arrays (forward)" << std::endl);
        FIXOUTVEC(config::Log,"Dimensions of FFT = ",n[0],n[1],n[2]);
        FIXOUT(config::Log,"rank (dimension of FFT) = " << 3 << std::endl);
        FIXOUT(config::Log,"How many (FFT's) = " << geom::ucm.GetNMS()*3 << std::endl);
        FIXOUTVEC(config::Log,"inembed = ",inembed[0],inembed[1],inembed[2]);
        FIXOUT(config::Log,"istride = " << istride << std::endl);
        FIXOUT(config::Log,"idist = " << idist << std::endl);
        FIXOUTVEC(config::Log,"onembed = ",onembed[0],onembed[1],onembed[2]);
        FIXOUT(config::Log,"ostride = " << ostride << std::endl);
        FIXOUT(config::Log,"odist = " << odist << std::endl);
        FIXOUT(config::Log,"Direction (sign) = " << "CUFFTW_FORWARD" << std::endl);
        if(hipfftPlanMany(&SPc2c,3,n,inembed,istride,idist,onembed,ostride,odist,HIPFFT_C2C,geom::ucm.GetNMS()*3)!=HIPFFT_SUCCESS)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("CUFFT 3D plan creation failed");
        }
        if(hipfftPlanMany(&FPc2c,3,n,onembed,ostride,odist,inembed,istride,idist,HIPFFT_C2C,geom::ucm.GetNMS()*3)!=HIPFFT_SUCCESS)
        {
            error::errPreamble(__FILE__,__LINE__);
            error::errMessage("CUFFT 3D plan creation failed");
        }


        //At this point we can copy the interaction matrix from the CPU
        //as there is no need to do the determination of the interaction
        //matrix on the card.
        //declare a holder on the heap
        Array7D<fftwf_complex> tempNkab;
        tempNkab.resize(geom::ucm.GetNMS(),geom::ucm.GetNMS(),3,3,geom::zpdim[0]*geom::Nk[0],geom::zpdim[1]*geom::Nk[1],geom::zpdim[2]*geom::Nk[2]);
        for(unsigned int s1 = 0 ; s1 < geom::ucm.GetNMS() ; s1++)
        {
            for(unsigned int s2 = 0 ; s2 < geom::ucm.GetNMS() ; s2++)
            {
                for(unsigned int alpha = 0 ; alpha < 3 ; alpha++)
                {
                    for(unsigned int beta = 0 ; beta < 3 ; beta++)
                    {
                        for(unsigned int i = 0 ; i < geom::zpdim[0]*geom::Nk[0] ; i++)
                        {
                            for(unsigned int j = 0 ; j < geom::zpdim[1]*geom::Nk[1] ; j++)
                            {
                                for(unsigned int k = 0 ; k < geom::zpdim[1]*geom::Nk[2] ; k++)
                                {
                                    for(unsigned int l = 0 ; l < 2 ; l++)
                                    {
                                        tempNkab(s1,s2,alpha,beta,i,j,k)[l]=static_cast<float>(intmat::Nkab(s1,s2,alpha,beta,i,j,k)[l]);
                                    }

                                }
                            }
                        }
                    }
                }
            }
        }

        //copy the FT'd interaction matrix to the card
        CUDA_CALL(hipMemcpy(CNk,tempNkab.ptr(),geom::ucm.GetNMS()*geom::ucm.GetNMS()*3*3*geom::zpdim[0]*geom::zpdim[1]*geom::zpdim[2]*geom::Nk[0]*geom::Nk[1]*geom::Nk[2]*sizeof(hipfftComplex),hipMemcpyHostToDevice));
        intmat::Nkab.clear();
        //clear the floating point holding arrays as well
        tempNkab.clear();
        check_cuda_errors(__FILE__,__LINE__);
    }
    void deallocate_cuda_memory()
    {
        config::printline(config::Info);
        config::Info.width(45);config::Info << std::right << "*" << "**EXIT information***" << std::endl;
        FIXOUT(config::Info,"Freeing space on GPU device" << std::flush);
        CUDA_CALL(hipFree(CNk));
        CUDA_CALL(hipFree(CSr));
        CUDA_CALL(hipFree(CSk));
        CUDA_CALL(hipFree(CHk));
        CUDA_CALL(hipFree(CHr));
        CUDA_CALL(hipFree(Cspin));
        CUDA_CALL(hipFree(Cespin));
        CUDA_CALL(hipFree(Crand));
        CUDA_CALL(hipFree(CH));
        CUDA_CALL(hipFree(Cfn));
        CUDA_CALL(hipFree(Csigma));
        CUDA_CALL(hipFree(Clambda));
        CUDA_CALL(hipFree(Cllgpf));
        CUDA_CALL(hipFree(Cspec));
        CUDA_CALL(hipFree(Ckx));
        CUDA_CALL(hipFree(Cky));
        CUDA_CALL(hipFree(Ckz));
        config::Info << "Done" << std::endl;
    }
    void spins_forward()
    {
        CUFFT_CALL(hipfftExecC2C(SPc2c,CSr,CSk,HIPFFT_FORWARD));
    }

    void fields_back()
    {
        CUFFT_CALL(hipfftExecC2C(FPc2c,CHk,CHr,HIPFFT_BACKWARD));
    }

    void allocate_memory_on_card()
    {
        //all of the GPU memory allocations should happen here.
        //--------------------------------------------------------------------------------
        CUDA_CALL(hipMalloc((void**)&CNk,geom::ucm.GetNMS()*geom::ucm.GetNMS()*3*3*geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CSk,geom::ucm.GetNMS()*3*geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CSr,geom::ucm.GetNMS()*3*geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CHk,geom::ucm.GetNMS()*3*geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&CHr,geom::ucm.GetNMS()*3*geom::zps*sizeof(hipfftComplex)));
        CUDA_CALL(hipMalloc((void**)&Cspin,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Cespin,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Crand,3*geom::nspins*sizeof(float)));
        CUDA_CALL(hipMalloc((void**)&CH,3*geom::nspins*sizeof(float)));
        CUDA_CALL(hipMalloc((void**)&Cfn,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Clambda,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Csigma,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Cllgpf,3*geom::nspins*sizeof(double)));
        CUDA_CALL(hipMalloc((void**)&Ckx,geom::nspins*sizeof(unsigned int)));
        CUDA_CALL(hipMalloc((void**)&Cky,geom::nspins*sizeof(unsigned int)));
        CUDA_CALL(hipMalloc((void**)&Ckz,geom::nspins*sizeof(unsigned int)));
        CUDA_CALL(hipMalloc((void**)&Cspec,geom::nspins*sizeof(unsigned int)));
        //--------------------------------------------------------------------------------
        //this section sorts out the copying of the data from the CPU to the card
        //--------------------------------------------------------------------------------
        //declare some arrays for doing copying to card
        //Nspins float array, 3*Nspins float array.
        float *nsfa=new float[geom::nspins];
        float *tnsfa=new float[3*geom::nspins];
        //Nspins double array, 3*Nspins double array
        double *nsda=new double[geom::nspins];
        double *tnsda=new double[3*geom::nspins];
        //Nspins int array, 3*Nspins int array
        int *nsia=new int[geom::nspins];
        //copy the location of the spins in real space to the device
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            nsia[i]=geom::lu(i,0);
        }
        CUDA_CALL(hipMemcpy(Ckx,nsia,geom::nspins*sizeof(unsigned int),hipMemcpyHostToDevice));
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            nsia[i]=geom::lu(i,1);
        }
        CUDA_CALL(hipMemcpy(Cky,nsia,geom::nspins*sizeof(unsigned int),hipMemcpyHostToDevice));
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            nsia[i]=geom::lu(i,2);
        }
        CUDA_CALL(hipMemcpy(Ckz,nsia,geom::nspins*sizeof(unsigned int),hipMemcpyHostToDevice));
        //and copy the species list
        for(unsigned int i = 0 ; i < geom::nspins ; i++)
        {
            nsia[i]=geom::lu(i,3);
        }
        CUDA_CALL(hipMemcpy(Cspec,nsia,geom::nspins*sizeof(unsigned int),hipMemcpyHostToDevice));
        int *tnsia=new int[3*geom::nspins];
        //copy spin data to single array
        util::copy3vecto1(geom::nspins,spins::Sx,spins::Sy,spins::Sz,tnsda);
        //copy spin data to card
        CUDA_CALL(hipMemcpy(Cspin,tnsda,3*geom::nspins*sizeof(double),hipMemcpyHostToDevice));
        //zero the field array
        for(unsigned int i = 0 ; i < 3*geom::nspins ; i++){tnsfa[i]=0.0;}CUDA_CALL(hipMemcpy(CH,tnsfa,3*geom::nspins*sizeof(float),hipMemcpyHostToDevice));
        //call the kernel to zero the spin array


        //make sure we clean up when the program exits
        atexit(deallocate_cuda_memory);
    }
}
