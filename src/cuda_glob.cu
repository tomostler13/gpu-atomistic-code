// File: cuda.cu
// Author:Tom Ostler
// Created: 26/06/2014
// Last-modified: 30 Sep 2014 16:16:56
#include "../inc/hip/hip_runtime.h"
#include "../inc/config.h"
#include "../inc/spins.h"
#include "../inc/mat.h"
#include "../inc/geom.h"
#include "../inc/config.h"
#include "../inc/random.h"
#include "../inc/intmat.h"
#include "../inc/util.h"
#include "../inc/fields.h"
#include "../inc/arrays.h"
#include "../inc/cudadefs.h"
#include "../inc/defines.h"
#include "../inc/cufields.h"
#include "../inc/cuint.h"
#include "../inc/llg.h"
//Cuda headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
//Library headers
#include <cmath>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <iostream>
namespace cullg
{
    hipDeviceProp_t deviceProp;
    hiprandGenerator_t gen;
    //number of threads per block and blocks per grid
    int threadsperblock,blockspergrid;
    //same but for the zero padded work spaces
    int zpblockspergrid;
    //same but for the complex zero padded work space (N_Z/2+1) for r2c and c2r transforms
    int czpblockspergrid;
    //rank of the FFT
    int nrank=3;
    //device pointers for Fourier space calculations
    hipfftComplex *CNk=NULL;
    hipfftComplex *CCSkx=NULL;
    hipfftComplex *CCSky=NULL;
    hipfftComplex *CCSkz=NULL;
    hipfftReal *CCSrx=NULL;
    hipfftReal *CCSry=NULL;
    hipfftReal *CCSrz=NULL;
    hipfftComplex *CCHkx=NULL;
    hipfftComplex *CCHky=NULL;
    hipfftComplex *CCHkz=NULL;
    hipfftReal *CCHrx=NULL;
    hipfftReal *CCHry=NULL;
    hipfftReal *CCHrz=NULL;

    //device pointers
    double *Cspin=NULL;
    double *Cespin=NULL;
    float *Crand=NULL;
    float *CH=NULL;
    int *Czpsn=NULL;//The is the zero pad spin number
    int *Clu=NULL;
    double *Cfn=NULL;
    //cufft plans
    hipfftHandle C3DPr2c,C3DPc2r;

}
