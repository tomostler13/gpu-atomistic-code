// File: cuda.cu
// Author:Tom Ostler
// Created: 26/06/2014
// Last-modified: 30 Sep 2014 19:31:54
#include "../inc/hip/hip_runtime.h"
#include "../inc/config.h"
#include "../inc/spins.h"
#include "../inc/mat.h"
#include "../inc/geom.h"
#include "../inc/config.h"
#include "../inc/random.h"
#include "../inc/intmat.h"
#include "../inc/util.h"
#include "../inc/fields.h"
#include "../inc/arrays.h"
#include "../inc/cudadefs.h"
#include "../inc/defines.h"
#include "../inc/cufields.h"
#include "../inc/cuint.h"
#include "../inc/llg.h"
//Cuda headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
//Library headers
#include <cmath>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <iostream>
namespace cullg
{
    hipDeviceProp_t deviceProp;
    hiprandGenerator_t gen;
    //number of threads per block and blocks per grid
    int threadsperblock,blockspergrid;
    //same but for the zero padded work spaces
    int zpblockspergrid;
    //same but for the complex zero padded work space (N_Z/2+1) for r2c and c2r transforms
    int czpblockspergrid;
    //rank of the FFT
    int nrank=3;
    //device pointers for Fourier space calculations
    hipfftComplex *CNk=NULL;
    hipfftComplex *CSk=NULL;
    hipfftReal *CSr=NULL;
    hipfftComplex *CHk=NULL;
    hipfftReal *CHr=NULL;
    //unsigned int the kx, ky and kz positions of the spins. The point is that you can use these arrays to
    //lookup which element of the array the the spin data should be copied to.
    unsigned int *Ckx=NULL,*Cky=NULL,*Ckz=NULL,*Cspec=NULL;

    //device pointers
    double *Cspin=NULL,*Cespin=NULL,*Crand=NULL;
    float *CH=NULL;
    double *Clambda=NULL,*Csigma=NULL,*Cfn=NULL,*Cllgpf=NULL;
    //cufft plans
    hipfftHandle C3DPr2c,C3DPc2r;

}
